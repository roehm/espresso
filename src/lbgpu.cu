#include "hip/hip_runtime.h"
/* 
   Copyright (C) 2010,2011,2012,2013 The ESPResSo project

   This file is part of ESPResSo.
  
   ESPResSo is free software: you can redistribute it and/or modify
   it under the terms of the GNU General Public License as published by
   the Free Software Foundation, either version 3 of the License, or
   (at your option) any later version.
   
   ESPResSo is distributed in the hope that it will be useful,
   but WITHOUT ANY WARRANTY; without even the implied warranty of
   MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
   GNU General Public License for more details.
   
   You should have received a copy of the GNU General Public License
   along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

/** \file lbgpu.cu
 *
 * Cuda (.cu) file for the Lattice Boltzmann implementation on GPUs.
 * Header file for \ref lbgpu.h.
 */

#include <stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

extern "C" {
#include "lbgpu.h"
#include "config.h"
//#include "cuda_common.h"
}

#ifdef LB_GPU
#ifndef GAUSSRANDOM
#define GAUSSRANDOM
#endif
#if 0
int extended_values_flag=0; /* TODO: this has to be set to one by
                               appropriate functions if there is 
                               the need to compute pi at every 
                               step (e.g. moving boundaries)*/

/**defining structures residing in global memory */

/** device_rho_v: struct for hydrodynamic fields: this is for internal use 
    (i.e. stores values in LB units) and should not used for 
    printing values  */
static LB_rho_v_gpu *device_rho_v= NULL;

/** device_rho_v_pi: extended struct for hydrodynamic fields: this is the interface
    to tcl, and stores values in MD units. It should not used
    as an input for any LB calculations. TODO: This structure is not yet 
    used, and it is here to allow access to the stress tensor at any
    timestep, e.g. for future implementations of moving boundary codes */
static LB_rho_v_gpu *device_rho_v_pi= NULL;

/** print_rho_v_pi: struct for hydrodynamic fields: this is the interface
    to tcl, and stores values in MD units. It should not used
    as an input for any LB calculations. TODO: in the future,
    one might want to have several structures for printing 
    separately rho, v, pi without having to compute/store 
    the complete set. */
static LB_rho_v_pi_gpu *print_rho_v_pi= NULL;

/** structs for velocity densities */
static LB_nodes_gpu nodes_a = {.vd=NULL,.seed=NULL,.boundary=NULL};
static LB_nodes_gpu nodes_b = {.vd=NULL,.seed=NULL,.boundary=NULL};;
/** struct for node force */
static LB_node_force_gpu node_f = {.force=NULL} ;

static LB_extern_nodeforce_gpu *extern_nodeforces = NULL;
#endif
#ifdef LB_BOUNDARIES_GPU
//static float* LB_boundary_force = NULL;
//static float* LB_boundary_velocity = NULL;
/** pointer for bound index array*/
static int *boundary_node_list;
static int *boundary_index_list;
static __device__ __constant__ int n_lb_boundaries_gpu = 0;
static size_t size_of_boundindex;
#endif

static unsigned int intflag = 1;
static LB_nodes_gpu *current_nodes = NULL;
/**defining size values for allocating global memory */
static size_t size_of_rho_v;
static size_t size_of_rho_v_pi;
static size_t size_of_forces;
static size_t size_of_particles;
static size_t size_of_seed;
static size_t size_of_extern_nodeforces;
static size_t size_of_uint;
static size_t size_of_nodes_gpu;
static size_t size_of_3floats; 
static size_t size_of_buffer[3];

/**parameters residing in constant memory */
static __device__ __constant__ LB_parameters_gpu para;
static __device__ __constant__ LB_gpus devpara;
static const float c_sound_sq = 1.f/3.f;

/**cudasteams for parallel computing on cpu and gpu */
hipStream_t *stream;
/**multi_gpu plan */
plan_gpu *plan;

//extern hipError_t err;
//extern hipError_t _err;
int plan_initflag = 0;

/*-------------------------------------------------------*/
/*********************************************************/
/** \name device functions called by kernel functions */
/*********************************************************/
/*-------------------------------------------------------*/

/*-------------------------------------------------------*/

/** atomic add function for sveral cuda architectures 
*/
__device__ inline void atomicadd(float* address, float value){
#if !defined __CUDA_ARCH__ || __CUDA_ARCH__ >= 200 // for Fermi, atomicAdd supports floats
  atomicAdd(address, value);
#elif __CUDA_ARCH__ >= 110
#warning Using slower atomicAdd emulation
// float-atomic-add from 
// [url="http://forums.nvidia.com/index.php?showtopic=158039&view=findpost&p=991561"]
  float old = value;
  while ((old = atomicExch(address, atomicExch(address, 0.0f)+old))!=0.0f);
#else
#error I need at least compute capability 1.1
#endif
}

/**randomgenerator which generates numbers [0,1]
 * @param *rn	Pointer to randomnumber array of the local node or particle 
*/
__device__ void random_01(LB_randomnr_gpu *rn){

  const float mxi = 1.f/(float)(1ul<<31);
  unsigned int curr = rn->seed;

  curr = 1103515245 * curr + 12345;
  rn->randomnr[0] = (float)(curr & ((1ul<<31)-1))*mxi;
  curr = 1103515245 * curr + 12345;
  rn->randomnr[1] = (float)(curr & ((1ul<<31)-1))*mxi;
  rn->seed = curr;

}

/** gaussian random nummber generator for thermalisation
 * @param *rn	Pointer to randomnumber array of the local node node or particle 
*/
__device__ void gaussian_random(LB_randomnr_gpu *rn){

  float x1, x2;
  float r2, fac;
  /** On every second call two gaussian random numbers are calculated
   via the Box-Muller transformation.*/
  /** draw two uniform random numbers in the unit circle */
  do {
    random_01(rn);
    x1 = 2.f*rn->randomnr[0]-1.f;
    x2 = 2.f*rn->randomnr[1]-1.f;
    r2 = x1*x1 + x2*x2;
  } while (r2 >= 1.f || r2 == 0.f);

  /** perform Box-Muller transformation */
  fac = sqrtf(-2.f*__logf(r2)/r2);
  rn->randomnr[0] = x2*fac;
  rn->randomnr[1] = x1*fac;
  
}
/* wrapper */
__device__ void random_wrapper(LB_randomnr_gpu *rn) { 

#ifdef GAUSSRANDOM
	gaussian_random(rn);	
#else 
#define sqrt12i 0.288675134594813f
        random_01(rn);
        rn->randomnr[0]-=0.5f;
        rn->randomnr[0]*=sqrt12i;
        rn->randomnr[1]-=0.5f;
        rn->randomnr[1]*=sqrt12i;
#endif   
}


/**tranformation from 1d array-index to xyz
 * @param index		node index / thread index (Input)
 * @param xyz		Pointer to calculated xyz array (Output)
 */
__device__ void index_to_xyz(unsigned int index, unsigned int *xyz){

  xyz[0] = index%para.dim_x;
  index /= para.dim_x;
  xyz[1] = index%para.dim_y;
  index /= para.dim_y;
  xyz[2] = index;
}

/**calculation of the modes from the velocitydensities (space-transform.)
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Output)
*/
__device__ void calc_m_from_n(LB_nodes_gpu n_a, unsigned int index, float *mode){
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
  /* mass mode */
  mode[0 + ii * LBQ] = n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index];

  /* momentum modes */
  mode[1 + ii * LBQ] = (n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
          + (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
          + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[2 + ii * LBQ] = (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
          + (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[3 + ii * LBQ] = (n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

  /* stress modes */
  mode[4 + ii * LBQ] = -(n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + index]) + n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]
          + n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index];
  mode[5 + ii * LBQ] = n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
          + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[6 + ii * LBQ] = (n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index])
          - 2.f*(n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
          - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] +n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]));
  mode[7 + ii * LBQ] = n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[8 + ii * LBQ] = n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[9 + ii * LBQ] = n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index] - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);

  /* kinetic modes */
  mode[10 + ii * LBQ] = -2.f*(n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[11 + ii * LBQ] = -2.f*(n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[12 + ii * LBQ] = -2.f*(n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[13 + ii * LBQ] = (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[14 + ii * LBQ] = (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[15 + ii * LBQ] = (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] - n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[16 + ii * LBQ] = n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]
           + n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index]
           + n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]
           - 2.f*((n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index]));
  mode[17 + ii * LBQ] = -(n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index]);
  mode[18 + ii * LBQ] = -(n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index])
           - (n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index]) - (n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index])
           + 2.f*((n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index]) + (n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index])
           + (n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index] + n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index]));

 }
}

__device__ void update_rho_v(float *mode, unsigned int index, LB_node_force_gpu node_f, LB_rho_v_gpu *d_v){

  float Rho_tot=0.f;
  float u_tot[3]={0.f,0.f,0.f};
  
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      /** re-construct the real density
      * remember that the populations are stored as differences to their
      * equilibrium value */
      d_v[index].rho[ii]= mode[0 + ii * LBQ]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
      Rho_tot  += mode[0 + ii * LBQ]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
      u_tot[0] += mode[1 + ii * LBQ];
      u_tot[1] += mode[2 + ii * LBQ];
      u_tot[2] += mode[3 + ii * LBQ];

      /** if forces are present, the momentum density is redefined to
      * inlcude one half-step of the force action.  See the
      * Chapman-Enskog expansion in [Ladd & Verberg]. */
      u_tot[0] += 0.5f*node_f.force[(0+ii*3)*para.number_of_nodes + index];
      u_tot[1] += 0.5f*node_f.force[(1+ii*3)*para.number_of_nodes + index];
      u_tot[2] += 0.5f*node_f.force[(2+ii*3)*para.number_of_nodes + index];
  }
  u_tot[0]/=Rho_tot;
  u_tot[1]/=Rho_tot;
  u_tot[2]/=Rho_tot;

  d_v[index].v[0]=u_tot[0]; 
  d_v[index].v[1]=u_tot[1]; 
  d_v[index].v[2]=u_tot[2]; 
}

/**lb_relax_modes, means collision update of the modes
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param node_f	Pointer to local node force (Input)
*/
__device__ void relax_modes(float *mode, unsigned int index, LB_node_force_gpu node_f, LB_rho_v_gpu *d_v){
  float u_tot[3]={0.f,0.f,0.f};

  update_rho_v(mode, index, node_f, d_v);
  u_tot[0]=d_v[index].v[0];  
  u_tot[1]=d_v[index].v[1];  
  u_tot[2]=d_v[index].v[2];  
 
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      float Rho; float j[3]; float pi_eq[6];

      Rho = mode[0 + ii * LBQ] + para.rho[ii]*para.agrid*para.agrid*para.agrid ;
      j[0] = Rho * u_tot[0];
      j[1] = Rho * u_tot[1];
      j[2] = Rho * u_tot[2];
      /** equilibrium part of the stress modes (eq13 schiller)*/

      pi_eq[0] = ((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2]))/Rho;
      pi_eq[1] = ((j[0]*j[0])-(j[1]*j[1]))/Rho;
      pi_eq[2] = (((j[0]*j[0])+(j[1]*j[1])+(j[2]*j[2])) - 3.0f*(j[2]*j[2]))/Rho;
      pi_eq[3] = j[0]*j[1]/Rho;
      pi_eq[4] = j[0]*j[2]/Rho;
      pi_eq[5] = j[1]*j[2]/Rho;
 
      /** in Shan-Chen we have to relax the momentum modes as well using the mobility, but
          the total momentum is conserved */  
#ifdef SHANCHEN
      mode[1 + ii * LBQ] = j[0] + para.gamma_mobility[0]*(mode[1 + ii * LBQ] - j[0]);
      mode[2 + ii * LBQ] = j[1] + para.gamma_mobility[0]*(mode[2 + ii * LBQ] - j[1]);
      mode[3 + ii * LBQ] = j[2] + para.gamma_mobility[0]*(mode[3 + ii * LBQ] - j[2]);
#endif
 
      /** relax the stress modes (eq14 schiller)*/
      mode[4 + ii * LBQ] = pi_eq[0] + para.gamma_bulk[ii]*(mode[4 + ii * LBQ] - pi_eq[0]);
      mode[5 + ii * LBQ] = pi_eq[1] + para.gamma_shear[ii]*(mode[5 + ii * LBQ] - pi_eq[1]);
      mode[6 + ii * LBQ] = pi_eq[2] + para.gamma_shear[ii]*(mode[6 + ii * LBQ] - pi_eq[2]);
      mode[7 + ii * LBQ] = pi_eq[3] + para.gamma_shear[ii]*(mode[7 + ii * LBQ] - pi_eq[3]);
      mode[8 + ii * LBQ] = pi_eq[4] + para.gamma_shear[ii]*(mode[8 + ii * LBQ] - pi_eq[4]);
      mode[9 + ii * LBQ] = pi_eq[5] + para.gamma_shear[ii]*(mode[9 + ii * LBQ] - pi_eq[5]);
    
      /** relax the ghost modes (project them out) */
      /** ghost modes have no equilibrium part due to orthogonality */
      mode[10 + ii * LBQ] = para.gamma_odd[ii]*mode[10 + ii * LBQ];
      mode[11 + ii * LBQ] = para.gamma_odd[ii]*mode[11 + ii * LBQ];
      mode[12 + ii * LBQ] = para.gamma_odd[ii]*mode[12 + ii * LBQ];
      mode[13 + ii * LBQ] = para.gamma_odd[ii]*mode[13 + ii * LBQ];
      mode[14 + ii * LBQ] = para.gamma_odd[ii]*mode[14 + ii * LBQ];
      mode[15 + ii * LBQ] = para.gamma_odd[ii]*mode[15 + ii * LBQ];
      mode[16 + ii * LBQ] = para.gamma_even[ii]*mode[16 + ii * LBQ];
      mode[17 + ii * LBQ] = para.gamma_even[ii]*mode[17 + ii * LBQ];
      mode[18 + ii * LBQ] = para.gamma_even[ii]*mode[18 + ii * LBQ];
 }
}


/**thermalization of the modes with gaussian random numbers
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param *rn		Pointer to randomnumber array of the local node
*/
__device__ void thermalize_modes(float *mode, unsigned int index, LB_randomnr_gpu *rn){
  float Rho;
#ifdef SHANCHEN
  random_wrapper(rn);
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      mode[1 + ii * LBQ] += sqrt((para.mu[ii]*(2.f/3.f)*(1.f-(para.gamma_mobility[0]*para.gamma_mobility[0])))) * (2*ii-1) * rn->randomnr[0];
      mode[2 + ii * LBQ] += sqrt((para.mu[ii]*(2.f/3.f)*(1.f-(para.gamma_mobility[0]*para.gamma_mobility[0])))) * (2*ii-1) * rn->randomnr[1];
  }
  random_wrapper(rn);
  for(int ii=0;ii<LB_COMPONENTS;++ii)  
      mode[3 + ii * LBQ] += sqrt((para.mu[ii]*(2.f/3.f)*(1.f-(para.gamma_mobility[0]*para.gamma_mobility[0])))) * (2*ii-1) * rn->randomnr[0];
#endif
  
  
  for(int ii=0;ii<LB_COMPONENTS;++ii) {  
      
      Rho = mode[0 + ii * LBQ] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
      /** momentum modes */
      random_wrapper(rn);
      /** stress modes */
      mode[4 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/3.f)*(1.f-(para.gamma_bulk[ii]*para.gamma_bulk[ii])))) * rn->randomnr[0];
      mode[5 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(4.f/9.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[6 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(4.f/3.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[0];
      mode[7 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(1.f/9.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[8 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(1.f/9.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[0];
      mode[9 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(1.f/9.f)*(1.f-(para.gamma_shear[ii]*para.gamma_shear[ii])))) * rn->randomnr[1];
      /** ghost modes */
      random_wrapper(rn);
      mode[10 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/3.f))) * rn->randomnr[0];
      mode[11 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/3.f))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[12 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/3.f))) * rn->randomnr[0];
      mode[13 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/9.f))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[14 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/9.f))) * rn->randomnr[0];
      mode[15 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f/9.f))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[16 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(2.f)))     * rn->randomnr[0];
      mode[17 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(4.f/9.f))) * rn->randomnr[1];
      random_wrapper(rn);
      mode[18 + ii * LBQ] += sqrt(Rho*(para.mu[ii]*(4.f/3.f))) * rn->randomnr[0];
   }
}


/*-------------------------------------------------------*/
/**normalization of the modes need befor backtransformation into velocity space
 * @param mode		Pointer to the local register values mode (Input/Output)
*/
__device__ void normalize_modes(float* mode){
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 

      /** normalization factors enter in the back transformation */
      mode[0 + ii * LBQ] *= 1.f;
      mode[1 + ii * LBQ] *= 3.f;
      mode[2 + ii * LBQ] *= 3.f;
      mode[3 + ii * LBQ] *= 3.f;
      mode[4 + ii * LBQ] *= 3.f/2.f;
      mode[5 + ii * LBQ] *= 9.f/4.f;
      mode[6 + ii * LBQ] *= 3.f/4.f;
      mode[7 + ii * LBQ] *= 9.f;
      mode[8 + ii * LBQ] *= 9.f;
      mode[9 + ii * LBQ] *= 9.f;
      mode[10 + ii * LBQ] *= 3.f/2.f;
      mode[11 + ii * LBQ] *= 3.f/2.f;
      mode[12 + ii * LBQ] *= 3.f/2.f;
      mode[13 + ii * LBQ] *= 9.f/2.f;
      mode[14 + ii * LBQ] *= 9.f/2.f;
      mode[15 + ii * LBQ] *= 9.f/2.f;
      mode[16 + ii * LBQ] *= 1.f/2.f;
      mode[17 + ii * LBQ] *= 9.f/4.f;
      mode[18 + ii * LBQ] *= 3.f/4.f;
  }
}



/*-------------------------------------------------------*/
/**backtransformation from modespace to desityspace and streaming with the push method using pbc
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param *n_b		Pointer to local node residing in array b (Output)
*/
__device__ void calc_n_from_modes_push(LB_nodes_gpu n_b, float *mode, unsigned int index){

  unsigned int xyz[3];
  index_to_xyz(index, xyz);
  unsigned int x = xyz[0];
  unsigned int y = xyz[1];
  unsigned int z = xyz[2];

  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 
  n_b.vd[(0 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/3.f * (mode[0 + ii * LBQ] - mode[4 + ii * LBQ] + mode[16 + ii * LBQ]);
  n_b.vd[(1 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] + mode[5 + ii * LBQ] + mode[6 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ] - 2.f*(mode[10 + ii * LBQ] + mode[16 + ii * LBQ]));
  n_b.vd[(2 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] + mode[5 + ii * LBQ] + mode[6 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ] + 2.f*(mode[10 + ii * LBQ] - mode[16 + ii * LBQ]));
  n_b.vd[(3 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0 + ii * LBQ] + mode[2 + ii * LBQ] - mode[5 + ii * LBQ] + mode[6 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ] - 2.f*(mode[11 + ii * LBQ] + mode[16 + ii * LBQ]));
  n_b.vd[(4 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/18.f * (mode[0 + ii * LBQ] - mode[2 + ii * LBQ] - mode[5 + ii * LBQ] + mode[6 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ] + 2.f*(mode[11 + ii * LBQ] - mode[16 + ii * LBQ]));
  n_b.vd[(5 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/18.f * (mode[0 + ii * LBQ] + mode[3 + ii * LBQ] - 2.f*(mode[6 + ii * LBQ] + mode[12 + ii * LBQ] + mode[16 + ii * LBQ] - mode[18 + ii * LBQ]));
  n_b.vd[(6 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/18.f * (mode[0 + ii * LBQ] - mode[3 + ii * LBQ] - 2.f*(mode[6 + ii * LBQ] - mode[12 + ii * LBQ] + mode[16 + ii * LBQ] - mode[18 + ii * LBQ]));
  n_b.vd[(7 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] + mode[2 + ii * LBQ] + mode[4 + ii * LBQ] + 2.f*mode[6 + ii * LBQ] + mode[7 + ii * LBQ] + mode[10 + ii * LBQ] + mode[11 + ii * LBQ] + mode[13 + ii * LBQ] + mode[14 + ii * LBQ] + mode[16 + ii * LBQ] + 2.f*mode[18 + ii * LBQ]);
  n_b.vd[(8 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] - mode[2 + ii * LBQ] + mode[4 + ii * LBQ] + 2.f*mode[6 + ii * LBQ] + mode[7 + ii * LBQ] - mode[10 + ii * LBQ] - mode[11 + ii * LBQ] - mode[13 + ii * LBQ] - mode[14 + ii * LBQ] + mode[16 + ii * LBQ] + 2.f*mode[18 + ii * LBQ]);
  n_b.vd[(9 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] - mode[2 + ii * LBQ] + mode[4 + ii * LBQ] + 2.f*mode[6 + ii * LBQ] - mode[7 + ii * LBQ] + mode[10 + ii * LBQ] - mode[11 + ii * LBQ] + mode[13 + ii * LBQ] - mode[14 + ii * LBQ] + mode[16 + ii * LBQ] + 2.f*mode[18 + ii * LBQ]);
  n_b.vd[(10 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] + mode[2 + ii * LBQ] + mode[4 + ii * LBQ] + 2.f*mode[6 + ii * LBQ] - mode[7 + ii * LBQ] - mode[10 + ii * LBQ] + mode[11 + ii * LBQ] - mode[13 + ii * LBQ] + mode[14 + ii * LBQ] + mode[16 + ii * LBQ] + 2.f*mode[18 + ii * LBQ]);
  n_b.vd[(11 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] + mode[3 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] - mode[6 + ii * LBQ] + mode[8 + ii * LBQ] + mode[10 + ii * LBQ] + mode[12 + ii * LBQ] - mode[13 + ii * LBQ] + mode[15 + ii * LBQ] + mode[16 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(12 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] - mode[3 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] - mode[6 + ii * LBQ] + mode[8 + ii * LBQ] - mode[10 + ii * LBQ] - mode[12 + ii * LBQ] + mode[13 + ii * LBQ] - mode[15 + ii * LBQ] + mode[16 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(13 + ii*LBQ ) * para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[1 + ii * LBQ] - mode[3 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] - mode[6 + ii * LBQ] - mode[8 + ii * LBQ] + mode[10 + ii * LBQ] - mode[12 + ii * LBQ] - mode[13 + ii * LBQ] - mode[15 + ii * LBQ] + mode[16 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(14 + ii*LBQ ) * para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[1 + ii * LBQ] + mode[3 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] - mode[6 + ii * LBQ] - mode[8 + ii * LBQ] - mode[10 + ii * LBQ] + mode[12 + ii * LBQ] + mode[13 + ii * LBQ] + mode[15 + ii * LBQ] + mode[16 + ii * LBQ] + mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(15 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[2 + ii * LBQ] + mode[3 + ii * LBQ] + mode[4 + ii * LBQ] - mode[5 + ii * LBQ] - mode[6 + ii * LBQ] + mode[9 + ii * LBQ] + mode[11 + ii * LBQ] + mode[12 + ii * LBQ] - mode[14 + ii * LBQ] - mode[15 + ii * LBQ] + mode[16 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(16 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[2 + ii * LBQ] - mode[3 + ii * LBQ] + mode[4 + ii * LBQ] - mode[5 + ii * LBQ] - mode[6 + ii * LBQ] + mode[9 + ii * LBQ] - mode[11 + ii * LBQ] - mode[12 + ii * LBQ] + mode[14 + ii * LBQ] + mode[15 + ii * LBQ] + mode[16 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(17 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] + mode[2 + ii * LBQ] - mode[3 + ii * LBQ] + mode[4 + ii * LBQ] - mode[5 + ii * LBQ] - mode[6 + ii * LBQ] - mode[9 + ii * LBQ] + mode[11 + ii * LBQ] - mode[12 + ii * LBQ] - mode[14 + ii * LBQ] + mode[15 + ii * LBQ] + mode[16 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);
  n_b.vd[(18 + ii*LBQ ) * para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = 1.f/36.f * (mode[0 + ii * LBQ] - mode[2 + ii * LBQ] + mode[3 + ii * LBQ] + mode[4 + ii * LBQ] - mode[5 + ii * LBQ] - mode[6 + ii * LBQ] - mode[9 + ii * LBQ] - mode[11 + ii * LBQ] + mode[12 + ii * LBQ] + mode[14 + ii * LBQ] - mode[15 + ii * LBQ] + mode[16 + ii * LBQ] - mode[17 + ii * LBQ] - mode[18 + ii * LBQ]);

}
}


#ifndef SHANCHEN
/*-------------------------------------------------------*/
/**backtransformation from modespace to desityspace and streaming with the push method and buffering of border desities
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param *n_b		Pointer to local node residing in array b (Output)
 * @param *buffer Pointer to buffer (Output)
*/
__device__ void calc_n_from_modes_buffer(LB_nodes_gpu n_b, float *buffer, float *mode, unsigned int index){

  unsigned int xyz[3];
  index_to_xyz(index, xyz);
  unsigned int x = xyz[0];
  unsigned int y = xyz[1];
  unsigned int z = xyz[2];
  //printf("x %i, y %i, z %i\n", x,y,z);
  unsigned nhyz = para.number_of_halo_nodes[0];
  unsigned nhxz = para.number_of_halo_nodes[1];
  unsigned nhxy = para.number_of_halo_nodes[2];
  //right buffered n's
  if(x == para.dim_x-1){
    //vd1
    buffer[0*nhyz + y + (para.dim_y*z)] = 1.f/18.f * (mode[0] + mode[1] + mode[5] + mode[6] - mode[17] - mode[18] - 2.f*(mode[10] + mode[16]));
    //vd7
    buffer[1*nhyz + y + (para.dim_y*z)] = 1.f/36.f * (mode[0] + mode[1] + mode[2] + mode[4] + 2.f*mode[6] + mode[7] + mode[10] + mode[11] + mode[13] + mode[14] + mode[16] + 2.f*mode[18]);
    //vd9
    buffer[2*nhyz + y + (para.dim_y*z)] = 1.f/36.f * (mode[0] + mode[1] - mode[2] + mode[4] + 2.f*mode[6] - mode[7] + mode[10] - mode[11] + mode[13] - mode[14] + mode[16] + 2.f*mode[18]);
    //vd11
    buffer[3*nhyz + y + (para.dim_y*z)] = 1.f/36.f * (mode[0] + mode[1] + mode[3] + mode[4] + mode[5] - mode[6] + mode[8] + mode[10] + mode[12] - mode[13] + mode[15] + mode[16] + mode[17] - mode[18]);
    //vd13
    buffer[4*nhyz + y + (para.dim_y*z)] = 1.f/36.f * (mode[0] + mode[1] - mode[3] + mode[4] + mode[5] - mode[6] - mode[8] + mode[10] - mode[12] - mode[13] - mode[15] + mode[16] + mode[17] - mode[18]);
    //non buffered n's
  }
  //left buffered n's
  if(x == 0){
    //vd2
    buffer[0*nhyz + y + (para.dim_y*z) + 5*nhyz] = 1.f/18.f * (mode[0] - mode[1] + mode[5] + mode[6] - mode[17] - mode[18] + 2.f*(mode[10] - mode[16]));
    //vd8
    buffer[1*nhyz + y + (para.dim_y*z) + 5*nhyz] = 1.f/36.f * (mode[0] - mode[1] - mode[2] + mode[4] + 2.f*mode[6] + mode[7] - mode[10] - mode[11] - mode[13] - mode[14] + mode[16] + 2.f*mode[18]);
    //vd10
    buffer[2*nhyz + y + (para.dim_y*z) + 5*nhyz] = 1.f/36.f * (mode[0] - mode[1] + mode[2] + mode[4] + 2.f*mode[6] - mode[7] - mode[10] + mode[11] - mode[13] + mode[14] + mode[16] + 2.f*mode[18]);
    //vd12
    buffer[3*nhyz + y + (para.dim_y*z) + 5*nhyz] = 1.f/36.f * (mode[0] - mode[1] - mode[3] + mode[4] + mode[5] - mode[6] + mode[8] - mode[10] - mode[12] + mode[13] - mode[15] + mode[16] + mode[17] - mode[18]);
    //vd14
    buffer[4*nhyz + y + (para.dim_y*z) + 5*nhyz] = 1.f/36.f * (mode[0] - mode[1] + mode[3] + mode[4] + mode[5] - mode[6] - mode[8] - mode[10] + mode[12] + mode[13] + mode[15] + mode[16] + mode[17] - mode[18]);
  }
  //back buffered n's
  if(y == (para.dim_y-1)){
    //vd3
    buffer[0*nhxz + x + (para.dim_x*z) + 2*5*nhyz] = 1.f/18.f * (mode[0] + mode[2] - mode[5] + mode[6] + mode[17] - mode[18] - 2.f*(mode[11] + mode[16]));
    //vd7
    buffer[1*nhxz + x + (para.dim_x*z) + 2*5*nhyz] = 1.f/36.f * (mode[0] + mode[1] + mode[2] + mode[4] + 2.f*mode[6] + mode[7] + mode[10] + mode[11] + mode[13] + mode[14] + mode[16] + 2.f*mode[18]);
    //vd10
    buffer[2*nhxz + x + (para.dim_x*z) + 2*5*nhyz] = 1.f/36.f * (mode[0] - mode[1] + mode[2] + mode[4] + 2.f*mode[6] - mode[7] - mode[10] + mode[11] - mode[13] + mode[14] + mode[16] + 2.f*mode[18]);
    //vd15
    buffer[3*nhxz + x + (para.dim_x*z) + 2*5*nhyz] = 1.f/36.f * (mode[0] + mode[2] + mode[3] + mode[4] - mode[5] - mode[6] + mode[9] + mode[11] + mode[12] - mode[14] - mode[15] + mode[16] - mode[17] - mode[18]);
    //vd17
    buffer[4*nhxz + x + (para.dim_x*z) + 2*5*nhyz] = 1.f/36.f * (mode[0] + mode[2] - mode[3] + mode[4] - mode[5] - mode[6] - mode[9] + mode[11] - mode[12] - mode[14] + mode[15] + mode[16] - mode[17] - mode[18]);
  }
  //front buffered n's
  if(y == 0){
    //vd4
    buffer[0*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)] = 1.f/18.f * (mode[0] - mode[2] - mode[5] + mode[6] + mode[17] - mode[18] + 2.f*(mode[11] - mode[16]));
    //vd8
    buffer[1*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)] = 1.f/36.f * (mode[0] - mode[1] - mode[2] + mode[4] + 2.f*mode[6] + mode[7] - mode[10] - mode[11] - mode[13] - mode[14] + mode[16] + 2.f*mode[18]);
    //vd9
    buffer[2*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)] = 1.f/36.f * (mode[0] + mode[1] - mode[2] + mode[4] + 2.f*mode[6] - mode[7] + mode[10] - mode[11] + mode[13] - mode[14] + mode[16] + 2.f*mode[18]);
    //vd16
    buffer[3*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)] = 1.f/36.f * (mode[0] - mode[2] - mode[3] + mode[4] - mode[5] - mode[6] + mode[9] - mode[11] - mode[12] + mode[14] + mode[15] + mode[16] - mode[17] - mode[18]);
    //vd18
    buffer[4*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)] = 1.f/36.f * (mode[0] - mode[2] + mode[3] + mode[4] - mode[5] - mode[6] - mode[9] - mode[11] + mode[12] + mode[14] - mode[15] + mode[16] - mode[17] - mode[18]);
  }
  //up buffered n's
  if(z == (para.dim_z-1)){
    //vd5
    buffer[0*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)] = 1.f/18.f * (mode[0] + mode[3] - 2.f*(mode[6] + mode[12] + mode[16] - mode[18]));
    //vd11
    buffer[1*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)] = 1.f/36.f * (mode[0] + mode[1] + mode[3] + mode[4] + mode[5] - mode[6] + mode[8] + mode[10] + mode[12] - mode[13] + mode[15] + mode[16] + mode[17] - mode[18]);
    //vd14
    buffer[2*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)] = 1.f/36.f * (mode[0] - mode[1] + mode[3] + mode[4] + mode[5] - mode[6] - mode[8] - mode[10] + mode[12] + mode[13] + mode[15] + mode[16] + mode[17] - mode[18]);
    //vd15
    buffer[3*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)] = 1.f/36.f * (mode[0] + mode[2] + mode[3] + mode[4] - mode[5] - mode[6] + mode[9] + mode[11] + mode[12] - mode[14] - mode[15] + mode[16] - mode[17] - mode[18]);
    //vd18
    buffer[4*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)] = 1.f/36.f * (mode[0] - mode[2] + mode[3] + mode[4] - mode[5] - mode[6] - mode[9] - mode[11] + mode[12] + mode[14] - mode[15] + mode[16] - mode[17] - mode[18]);
  }
  //down buffered n's
  if(z == 0){
    //vd6
    buffer[0*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)] = 1.f/18.f * (mode[0] - mode[3] - 2.f*(mode[6] - mode[12] + mode[16] - mode[18]));
    //vd12
    buffer[1*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)] = 1.f/36.f * (mode[0] - mode[1] - mode[3] + mode[4] + mode[5] - mode[6] + mode[8] - mode[10] - mode[12] + mode[13] - mode[15] + mode[16] + mode[17] - mode[18]);
    //vd13
    buffer[2*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)] = 1.f/36.f * (mode[0] + mode[1] - mode[3] + mode[4] + mode[5] - mode[6] - mode[8] + mode[10] - mode[12] - mode[13] - mode[15] + mode[16] + mode[17] - mode[18]);
    //vd16
    buffer[3*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)] = 1.f/36.f * (mode[0] - mode[2] - mode[3] + mode[4] - mode[5] - mode[6] + mode[9] - mode[11] - mode[12] + mode[14] + mode[15] + mode[16] - mode[17] - mode[18]);
    //vd17
    buffer[4*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)] = 1.f/36.f * (mode[0] + mode[2] - mode[3] + mode[4] - mode[5] - mode[6] - mode[9] + mode[11] - mode[12] - mode[14] + mode[15] + mode[16] - mode[17] - mode[18]);
  }
}

/*-------------------------------------------------------*/
/**write the received vds from buffer into their correct position in the nodes array 
 * @param index		node index / thread index (Input)
 * @param *n_b		Pointer to local node residing in array b (Output)
 * @param *buffer Pointer to buffer (Input)
 */
__device__ void write_n_from_buffer(LB_nodes_gpu n_b, float *buffer, unsigned int index){

  unsigned int xyz[3];
  index_to_xyz(index, xyz);
  unsigned int x = xyz[0];
  unsigned int y = xyz[1];
  unsigned int z = xyz[2];
  unsigned nhyz = para.number_of_halo_nodes[0];
  unsigned nhxz = para.number_of_halo_nodes[1];
  unsigned nhxy = para.number_of_halo_nodes[2];
  //right
  if(x == 0){
    //vd1
    n_b.vd[1*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = buffer[0*nhyz + y + (para.dim_y*z)];
    //vd7
    n_b.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[1*nhyz + y + (para.dim_y*z)];
    //vd9
    n_b.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[2*nhyz + y + (para.dim_y*z)];
    //vd11
    n_b.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = buffer[3*nhyz + y + (para.dim_y*z)];
    //vd13
    n_b.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = buffer[4*nhyz + y + (para.dim_y*z)];
  }
  //left
  if(x == (para.dim_x-1)){
    //vd2
    n_b.vd[2*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = buffer[0*nhyz + y + (para.dim_y*z) + 5*nhyz];
    //vd8
    n_b.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[1*nhyz + y + (para.dim_y*z) + 5*nhyz];
    //vd10
    n_b.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[2*nhyz + y + (para.dim_y*z) + 5*nhyz];
    //vd12
    n_b.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = buffer[3*nhyz + y + (para.dim_y*z) + 5*nhyz];
    //vd14
    n_b.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = buffer[4*nhyz + y + (para.dim_y*z) + 5*nhyz];
  }
  //back
  if(y == 0){
    //vd3
    n_b.vd[3*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[0*nhxz + x + (para.dim_x*z) + 5*2*nhyz];
    //vd7
    n_b.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[1*nhxz + x + (para.dim_x*z) + 5*2*nhyz];
    //vd10
    n_b.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[2*nhxz + x + (para.dim_x*z) + 5*2*nhyz];
    //vd15
    n_b.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = buffer[3*nhxz + x + (para.dim_x*z) + 5*2*nhyz];
    //vd17
    n_b.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = buffer[4*nhxz + x + (para.dim_x*z) + 5*2*nhyz];
  }
  //front
  if(y == (para.dim_y-1)){
    //vd4
    n_b.vd[4*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[0*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)];
    //vd8
    n_b.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[1*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)];
    //vd9
    n_b.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = buffer[2*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)];
    //vd16
    n_b.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = buffer[3*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)];
    //vd18
    n_b.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = buffer[4*nhxz + x + (para.dim_x*z) + 5*(2*nhyz + nhxz)];
  }
  //up
  if(z == 0){
    //vd5
    n_b.vd[5*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = buffer[0*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)];
    //vd11
    n_b.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = buffer[1*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)];
    //vd14
    n_b.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = buffer[2*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)];
    //vd15
    n_b.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = buffer[3*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)];
    //vd18
    n_b.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = buffer[4*nhxy + x + (para.dim_x*y) + 5*2*(nhyz + nhxz)];
  }
  //down
  if(z == (para.dim_z-1)){
    //vd6
    n_b.vd[6*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = buffer[0*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)];
    //vd12
    n_b.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = buffer[1*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)];
    //vd13
    n_b.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = buffer[2*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)];
    //vd16
    n_b.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = buffer[3*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)];
    //vd17
    n_b.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = buffer[4*nhxy + x + (para.dim_x*y) + 5*(2*(nhyz + nhxz) + nhxy)];
  }
}
/** Bounce back boundary conditions.
 * The populations that have propagated into a boundary node
 * are bounced back to the node they came from. This results
 * in no slip boundary conditions.
 *
 * [cf. Ladd and Verberg, J. Stat. Phys. 104(5/6):1191-1251, 2001]
 * @param index			node index / thread index (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
 * @param n_a			Pointer to local node residing in array a (Output) (temp stored in buffer a)
 * @param LB_boundary_velocity 			The constant velocity at the boundary, set by the user (Input)
 * @param LB_boundary_force 			The force on the boundary nodes (Output)
*/
__device__ void bounce_back_read(LB_nodes_gpu n_b, LB_nodes_gpu n_a, unsigned int index, \
    float* LB_boundary_velocity, float* LB_boundary_force){
    
  unsigned int xyz[3];
  int c[3];
  float v[3];
  float shift, weight, pop_to_bounce_back;
  float boundary_force[3] = {0,0,0};
  size_t to_index, to_index_x, to_index_y, to_index_z;
  int population, inverse;
  int boundary_index;


  boundary_index=n_b.boundary[index];
  if(boundary_index != 0){
    
    v[0]=LB_boundary_velocity[3*(boundary_index-1)+0];
    v[1]=LB_boundary_velocity[3*(boundary_index-1)+1];
    v[2]=LB_boundary_velocity[3*(boundary_index-1)+2];

    index_to_xyz(index, xyz);

    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

/* CPU analog of shift:
   lbpar.agrid*lbpar.agrid*lbpar.agrid*lbpar.rho*2*lbmodel.c[i][l]*lb_boundaries[lbfields[k].boundary-1].velocity[l] */
  
    /** store vd temporary in second lattice to avoid race conditions */
   // TODO: fix the multicomponent version (rho...)
#define BOUNCEBACK  \
  shift = para.agrid*para.agrid*para.agrid*para.agrid*para.rho[0]*2.*3.*weight*para.tau*(v[0]*c[0] + v[1]*c[1] + v[2]*c[2]); \
  pop_to_bounce_back = n_b.vd[population*para.number_of_nodes + index ]; \
  to_index_x = (x+c[0]+para.dim_x)%para.dim_x; \
  to_index_y = (y+c[1]+para.dim_y)%para.dim_y; \
  to_index_z = (z+c[2]+para.dim_z)%para.dim_z; \
  to_index = to_index_x + para.dim_x*to_index_y + para.dim_x*para.dim_y*to_index_z; \
  if (n_b.boundary[to_index] == 0) \
  { \
    boundary_force[0] += (2*pop_to_bounce_back+shift)*c[0]/para.tau/para.tau/para.agrid; \
    boundary_force[1] += (2*pop_to_bounce_back+shift)*c[1]/para.tau/para.tau/para.agrid; \
    boundary_force[2] += (2*pop_to_bounce_back+shift)*c[2]/para.tau/para.tau/para.agrid; \
    n_b.vd[inverse*para.number_of_nodes + to_index ] = pop_to_bounce_back + shift; \
  }

// ***** SHOULDN'T THERE BE AN ELSE STATMENT IN "BOUNCEBACK"?
// ***** THERE IS AN ODD FACTOR OF 2 THAT YOU INCUR IN THE FORCES FOR THE "lb_stokes_sphere_gpu.tcl" TEST CASE

    // the resting population does nothing.
    c[0]=1;c[1]=0;c[2]=0; weight=1./18.; population=2; inverse=1; 
    BOUNCEBACK
    
    c[0]=-1;c[1]=0;c[2]=0; weight=1./18.; population=1; inverse=2; 
    BOUNCEBACK
    
    c[0]=0;c[1]=1;c[2]=0;  weight=1./18.; population=4; inverse=3; 
    BOUNCEBACK

    c[0]=0;c[1]=-1;c[2]=0; weight=1./18.; population=3; inverse=4; 
    BOUNCEBACK
    
    c[0]=0;c[1]=0;c[2]=1; weight=1./18.; population=6; inverse=5; 
    BOUNCEBACK

    c[0]=0;c[1]=0;c[2]=-1; weight=1./18.; population=5; inverse=6; 
    BOUNCEBACK 
    
    c[0]=1;c[1]=1;c[2]=0; weight=1./36.; population=8; inverse=7; 
    BOUNCEBACK
    
    c[0]=-1;c[1]=-1;c[2]=0; weight=1./36.; population=7; inverse=8; 
    BOUNCEBACK
    
    c[0]=1;c[1]=-1;c[2]=0; weight=1./36.; population=10; inverse=9; 
    BOUNCEBACK

    c[0]=-1;c[1]=+1;c[2]=0; weight=1./36.; population=9; inverse=10; 
    BOUNCEBACK
    
    c[0]=1;c[1]=0;c[2]=1; weight=1./36.; population=12; inverse=11; 
    BOUNCEBACK
    
    c[0]=-1;c[1]=0;c[2]=-1; weight=1./36.; population=11; inverse=12; 
    BOUNCEBACK

    c[0]=1;c[1]=0;c[2]=-1; weight=1./36.; population=14; inverse=13; 
    BOUNCEBACK
    
    c[0]=-1;c[1]=0;c[2]=1; weight=1./36.; population=13; inverse=14; 
    BOUNCEBACK

    c[0]=0;c[1]=1;c[2]=1; weight=1./36.; population=16; inverse=15; 
    BOUNCEBACK
    
    c[0]=0;c[1]=-1;c[2]=-1; weight=1./36.; population=15; inverse=16; 
    BOUNCEBACK
    
    c[0]=0;c[1]=1;c[2]=-1; weight=1./36.; population=18; inverse=17; 
    BOUNCEBACK
    
    c[0]=0;c[1]=-1;c[2]=1; weight=1./36.; population=17; inverse=18; 
    BOUNCEBACK  
    
    atomicadd(&LB_boundary_force[3*(n_b.boundary[index]-1)+0], boundary_force[0]);
    atomicadd(&LB_boundary_force[3*(n_b.boundary[index]-1)+1], boundary_force[1]);
    atomicadd(&LB_boundary_force[3*(n_b.boundary[index]-1)+2], boundary_force[2]);
  }
}


#else  // SHANCHEN

// To be implemented


#endif // SHANCHEN

#ifndef SHANCHEN

/**bounce back read kernel needed to avoid raceconditions
 * @param index			node index / thread index (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
 * @param n_a			Pointer to local node residing in array a (Output) (temp stored in buffer a)
*/
__device__ void bounce_back_write(LB_nodes_gpu n_b, LB_nodes_gpu n_a, unsigned int index){

  unsigned int xyz[3];

  if(n_b.boundary[index] != 0){
    index_to_xyz(index, xyz);
    unsigned int x = xyz[0];
    unsigned int y = xyz[1];
    unsigned int z = xyz[2];

    /** stream vd from boundary node back to origin node */
    n_b.vd[1*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_a.vd[1*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z];
    n_b.vd[2*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z] = n_a.vd[2*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z];
    n_b.vd[3*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[3*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[4*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[4*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[5*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[5*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[6*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[6*para.number_of_nodes + x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[7*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[8*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[9*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z] = n_a.vd[10*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z];
    n_b.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[11*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[12*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[13*para.number_of_nodes + (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[14*para.number_of_nodes + (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[15*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
    n_b.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[16*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)] = n_a.vd[17*para.number_of_nodes + x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z)];
    n_b.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)] = n_a.vd[18*para.number_of_nodes + x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z)];
  }
}

#else // SHANCHEN

// to be implemented

#endif // SHANCHEN


/** add of (external) forces within the modespace, needed for particle-interaction
 * @param index		node index / thread index (Input)
 * @param mode		Pointer to the local register values mode (Input/Output)
 * @param node_f	Pointer to local node force (Input)
*/
__device__ void apply_forces(unsigned int index, float *mode, LB_node_force_gpu node_f, LB_rho_v_gpu *d_v) {
  
  float u[3]={0.f,0.f,0.f}, C[6]={0.f,0.f,0.f,0.f,0.f,0.f};
  float force_factor=powf(para.agrid,4)*para.tau*para.tau;
  /* Note: the values d_v were calculated in relax_modes() */

  u[0]=d_v[index].v[0]; 
  u[1]=d_v[index].v[1]; 
  u[2]=d_v[index].v[2]; 


  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) {  
       C[0] += (1.f + para.gamma_bulk[ii])*u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] + 
                1.f/3.f*(para.gamma_bulk[ii]-para.gamma_shear[ii])*(u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] + 
                        u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] + 
                        u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]);
       C[2] += (1.f + para.gamma_bulk[ii])*u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] + 
                1.f/3.f*(para.gamma_bulk[ii]-para.gamma_shear[ii])*(u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] + 
                        u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] + 
                        u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]);
       C[5] += (1.f + para.gamma_bulk[ii])*u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] + 
                1.f/3.f*(para.gamma_bulk[ii]-para.gamma_shear[ii])*(u[0]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] + 
                        u[1]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] + 
                        u[2]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]);
       C[1] += 1.f/2.f*(1.f+para.gamma_shear[ii])*(u[0]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index]+
                        u[1]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index]);
       C[3] += 1.f/2.f*(1.f+para.gamma_shear[ii])*(u[0]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]+
                        u[2]*node_f.force[(0 + ii*3 ) * para.number_of_nodes + index]);
       C[4] += 1.f/2.f*(1.f+para.gamma_shear[ii])*(u[1]*node_f.force[(2 + ii*3 ) * para.number_of_nodes + index]+
                        u[2]*node_f.force[(1 + ii*3 ) * para.number_of_nodes + index]);
  }

  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) {  
      /** update momentum modes */
#ifdef SHANCHEN
      float mobility_factor=1.f/2.f*(1.f+para.gamma_mobility[0]);
#else
      float mobility_factor=1.f;
#endif 
 /** update momentum modes */
      mode[1 + ii * LBQ] += mobility_factor * node_f.force[(0 + ii*3 ) * para.number_of_nodes + index];
      mode[2 + ii * LBQ] += mobility_factor * node_f.force[(1 + ii*3 ) * para.number_of_nodes + index];
      mode[3 + ii * LBQ] += mobility_factor * node_f.force[(2 + ii*3 ) * para.number_of_nodes + index];
      	
      /** update stress modes */
      mode[4 + ii * LBQ] += C[0] + C[2] + C[5];
      mode[5 + ii * LBQ] += C[0] - C[2];
      mode[6 + ii * LBQ] += C[0] + C[2] - 2.f*C[5];
      mode[7 + ii * LBQ] += C[1];
      mode[8 + ii * LBQ] += C[3];
      mode[9 + ii * LBQ] += C[4];
    
#ifdef EXTERNAL_FORCES
      if(para.external_force){
        node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] = para.ext_force[0]*force_factor;
        node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] = para.ext_force[1]*force_factor;
        node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] = para.ext_force[2]*force_factor;
      }
      else{
      node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      }
#else
      /** reset force */
      node_f.force[(0 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      node_f.force[(1 + ii*3 ) * para.number_of_nodes + index] = 0.f;
      node_f.force[(2 + ii*3 ) * para.number_of_nodes + index] = 0.f;
#endif
  }
}

/**function used to calculate hydrodynamic fields in MD units.
 * @param n_a		Pointer to local node residing in array a for boundary flag(Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param d_p_v         Pointer to local print values (Output)
 * @param d_v           Pointer to local device values (Input)
 * @param index		node index / thread index (Input)
*/
__device__ void calc_values_in_MD_units(LB_nodes_gpu n_a, float *mode,  LB_rho_v_pi_gpu *d_p_v, LB_rho_v_gpu * d_v, unsigned int index, unsigned int print_index) {
  
  float j[3]; 
  float pi_eq[6] ; 
  float pi[6]={0.f,0.f,0.f,0.f,0.f,0.f};
  float rho_tot=0.f;

  if(n_a.boundary[index] == 0) {

    for(int ii= 0; ii < LB_COMPONENTS; ii++) {
   	  rho_tot += d_v[index].rho[ii];
      d_p_v[print_index].rho[ii] = d_v[index].rho[ii] / para.agrid / para.agrid / para.agrid;
    }
      
    d_p_v[print_index].v[0] = d_v[index].v[0] / para.tau / para.agrid;
    d_p_v[print_index].v[1] = d_v[index].v[1] / para.tau / para.agrid;
    d_p_v[print_index].v[2] = d_v[index].v[2] / para.tau / para.agrid;

    /* stress calculation */ 
    for(int ii = 0; ii < LB_COMPONENTS; ii++) {
      float Rho = d_v[index].rho[ii];
      
      /* note that d_v[index].v[] already includes the 1/2 f term, accounting for the pre- and post-collisional average */
      j[0] = Rho * d_v[index].v[0];
      j[1] = Rho * d_v[index].v[1];
      j[2] = Rho * d_v[index].v[2];
      
      /* equilibrium part of the stress modes */
      pi_eq[0] = ( j[0]*j[0] + j[1]*j[1] + j[2]*j[2] ) / Rho;
      pi_eq[1] = ( j[0]*j[0] - j[1]*j[1] )/ Rho;
      pi_eq[2] = ( j[0]*j[0] + j[1]*j[1] + j[2]*j[2] - 3.0*j[2]*j[2] ) / Rho;
      pi_eq[3] = j[0]*j[1] / Rho;
      pi_eq[4] = j[0]*j[2] / Rho;
      pi_eq[5] = j[1]*j[2] / Rho;
     
      /* Now we must predict the outcome of the next collision */
      /* We immediately average pre- and post-collision.  */
      /* TODO: need a reference for this.   */
      mode[4 + ii * LBQ ] = pi_eq[0] + (0.5 + 0.5*para.gamma_bulk[ii] ) * (mode[4 + ii * LBQ] - pi_eq[0]);
      mode[5 + ii * LBQ ] = pi_eq[1] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[5 + ii * LBQ] - pi_eq[1]);
      mode[6 + ii * LBQ ] = pi_eq[2] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[6 + ii * LBQ] - pi_eq[2]);
      mode[7 + ii * LBQ ] = pi_eq[3] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[7 + ii * LBQ] - pi_eq[3]);
      mode[8 + ii * LBQ ] = pi_eq[4] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[8 + ii * LBQ] - pi_eq[4]);
      mode[9 + ii * LBQ ] = pi_eq[5] + (0.5 + 0.5*para.gamma_shear[ii]) * (mode[9 + ii * LBQ] - pi_eq[5]);
     
      /* Now we have to transform to the "usual" stress tensor components */
      /* We use eq. 116ff in Duenweg Ladd for that. */
      pi[0] += ( mode[0 + ii * LBQ] + mode[4 + ii * LBQ] + mode[5 + ii * LBQ] ) / 3.0;
      pi[2] += ( 2*mode[0 + ii * LBQ] + 2*mode[4 + ii * LBQ] - mode[5 + ii * LBQ] + 3*mode[6 + ii * LBQ] ) / 6.;
      pi[5] += ( 2*mode[0 + ii * LBQ] + 2*mode[4 + ii * LBQ] - mode[5 + ii * LBQ] + 3*mode[6 + ii * LBQ ]) / 6.;
      pi[1] += mode[7 + ii * LBQ];
      pi[3] += mode[8 + ii * LBQ];
      pi[4] += mode[9 + ii * LBQ];
    }
     
    for(int i = 0; i < 6; i++) {
      d_p_v[print_index].pi[i] = pi[i]  /para.tau / para.tau / para.agrid / para.agrid / para.agrid;
    }
  }
  else {
    for(int ii = 0; ii < LB_COMPONENTS; ii++)
	    d_p_v[print_index].rho[ii] = 0.0f;
     
    for(int i = 0; i < 3; i++)
     	d_p_v[print_index].v[i] = 0.0f;
     	
    for(int i = 0; i < 6; i++)
     	d_p_v[print_index].pi[i] = 0.0f;
  }
}

/**function used to calc physical values of every node
 * @param n_a		Pointer to local node residing in array a for boundary flag(Input)
 * @param mode		Pointer to the local register values mode (Input)
 * @param d_v		Pointer to local device values (Input/Output)
 * @param index		node index / thread index (Input)
*/

/* FIXME this function is basically un-used, think about removing/replacing it */
__device__ void calc_values(LB_nodes_gpu n_a, float *mode, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f, unsigned int index){ 

  float Rho_tot=0.f;
  float u_tot[3]={0.f,0.f,0.f};

  if(n_a.boundary[index] != 1){
      #pragma unroll
      for(int ii=0;ii<LB_COMPONENTS;++ii) { 
          /** re-construct the real density
          * remember that the populations are stored as differences to their
          * equilibrium value */
          d_v[index].rho[ii]= mode[0 + ii * 4]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
          Rho_tot  += mode[0 + ii * 4]+ para.rho[ii]*para.agrid*para.agrid*para.agrid;
          u_tot[0] += mode[1 + ii * 4];
          u_tot[1] += mode[2 + ii * 4];
          u_tot[2] += mode[3 + ii * 4];
    
          /** if forces are present, the momentum density is redefined to
          * inlcude one half-step of the force action.  See the
          * Chapman-Enskog expansion in [Ladd & Verberg]. */
    
          u_tot[0] += 0.5f*node_f.force[(0+ii*3)*para.number_of_nodes + index];
          u_tot[1] += 0.5f*node_f.force[(1+ii*3)*para.number_of_nodes + index];
          u_tot[2] += 0.5f*node_f.force[(2+ii*3)*para.number_of_nodes + index];
      }
      u_tot[0]/=Rho_tot;
      u_tot[1]/=Rho_tot;
      u_tot[2]/=Rho_tot;
    
      d_v[index].v[0]=u_tot[0]; 
      d_v[index].v[1]=u_tot[1]; 
      d_v[index].v[2]=u_tot[2]; 
  } else { 
    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;++ii) { 
       d_v[index].rho[ii]   = 1.;
    }
    d_v[index].v[0] = 0.;
    d_v[index].v[1] = 0.; 
    d_v[index].v[2] = 0.; 
  }   
}


/** 
 * @param node_index	node index around (8) particle (Input)
 * @param *mode			Pointer to the local register values mode (Output)
 * @param n_a			Pointer to local node residing in array a(Input)
*/
__device__ void calc_mode(float *mode, LB_nodes_gpu n_a, unsigned int node_index, int component_index){
	
  /** mass mode */
  mode[0] = n_a.vd[(0 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(1 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(2 + component_index*LBQ ) * para.number_of_nodes + node_index] 
          + n_a.vd[(3 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(4 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(5 + component_index*LBQ ) * para.number_of_nodes + node_index]
          + n_a.vd[(6 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(7 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(8 + component_index*LBQ ) * para.number_of_nodes + node_index]
          + n_a.vd[(9 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + node_index]
          + n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + node_index]
          + n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + node_index] + n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + node_index];

  /** momentum modes */
  mode[1] = (n_a.vd[(1 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(2 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(7 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(8 + component_index*LBQ ) * para.number_of_nodes + node_index])
          + (n_a.vd[(9 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + node_index])
          + (n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + node_index]);
  mode[2] = (n_a.vd[(3 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(4 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(7 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(8 + component_index*LBQ ) * para.number_of_nodes + node_index])
          - (n_a.vd[(9 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + node_index])
          + (n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + node_index]);
  mode[3] = (n_a.vd[(5 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(6 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + node_index])
          - (n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + node_index]) + (n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + node_index])
          - (n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + node_index] - n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + node_index]);
}


/**calculate temperature of the fluid kernel
 * @param *cpu_jsquared			Pointer to result storage value (Output)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void temperature(LB_nodes_gpu n_a, float *cpu_jsquared) {
  float mode[4];
  float jsquared = 0.f;
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    if(!n_a.boundary[index]){
     for(int ii=0;ii<LB_COMPONENTS;++ii) {  
         calc_mode(mode, n_a, index,ii);
         jsquared = mode[1]*mode[1]+mode[2]*mode[2]+mode[3]*mode[3];
         atomicadd(cpu_jsquared, jsquared);
     }
   }
 }
}


/*********************************************************/
/** \name Coupling part */
/*********************************************************/
/**(Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param n_a			Pointer to local node residing in array a (Input)
 * @param *delta		Pointer for the weighting of particle position (Output)
 * @param *delta_j		Pointer for the weighting of particle momentum (Output)
 * @param *particle_data	Pointer to the particle position and velocity (Input)
 * @param *particle_force	Pointer to the particle force (Input)
 * @param part_index		particle id / thread id (Input)
 * @param *rn_part		Pointer to randomnumber array of the particle
 * @param node_index		node index around (8) particle (Output)
*/
__device__ void calc_viscous_force(LB_nodes_gpu n_a, float *delta, float * partgrad1, float * partgrad2, float * partgrad3, CUDA_particle_data *particle_data, CUDA_particle_force *particle_force, unsigned int part_index, LB_randomnr_gpu *rn_part, float *delta_j, unsigned int *node_index, LB_rho_v_gpu *d_v){
	
 int my_left[3];
 float interpolated_u1, interpolated_u2, interpolated_u3;
 float interpolated_rho[LB_COMPONENTS];
 float temp_delta[6];
 float temp_delta_half[6];
 float viscforce[3*LB_COMPONENTS];
 float scforce[3*LB_COMPONENTS];
 float mode[19*LB_COMPONENTS];
#ifdef SHANCHEN
 float gradrho1, gradrho2, gradrho3;
 float Rho;
#endif 

 #pragma unroll
 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
   #pragma unroll
   for(int jj=0; jj<3; ++jj){ 
    scforce[jj+ii*3]  =0.f;
    viscforce[jj+ii*3]=0.f;
    delta_j[jj+ii*3]  =0.f;
   }
   #pragma unroll
   for(int jj=0; jj<8; ++jj){ 
    partgrad1[jj+ii*8]=0.f;
    partgrad2[jj+ii*8]=0.f;
    partgrad3[jj+ii*8]=0.f;
   }
 }
 /** see ahlrichs + duenweg page 8227 equ (10) and (11) */
 #pragma unroll
 for(int i=0; i<3; ++i){
   float scaledpos = particle_data[part_index].p[i]/para.agrid - 0.5f;
   my_left[i] = (int)(floorf(scaledpos));
   //printf("scaledpos %f \t myleft: %d \n", scaledpos, my_left[i]);
   temp_delta[3+i] = scaledpos - my_left[i];
   temp_delta[i] = 1.f - temp_delta[3+i];
   /**further value used for interpolation of fluid velocity at part pos near boundaries */
   temp_delta_half[3+i] = (scaledpos - my_left[i])*2.f;
   temp_delta_half[i] = 2.f - temp_delta_half[3+i];
 }

 delta[0] = temp_delta[0] * temp_delta[1] * temp_delta[2];
 delta[1] = temp_delta[3] * temp_delta[1] * temp_delta[2];
 delta[2] = temp_delta[0] * temp_delta[4] * temp_delta[2];
 delta[3] = temp_delta[3] * temp_delta[4] * temp_delta[2];
 delta[4] = temp_delta[0] * temp_delta[1] * temp_delta[5];
 delta[5] = temp_delta[3] * temp_delta[1] * temp_delta[5];
 delta[6] = temp_delta[0] * temp_delta[4] * temp_delta[5];
 delta[7] = temp_delta[3] * temp_delta[4] * temp_delta[5];

 // modulo for negative numbers is strange at best, shift to make sure we are positive
 int x = my_left[0] + para.dim_x;
 int y = my_left[1] + para.dim_y;
 int z = my_left[2] + para.dim_z;

 node_index[0] = x%para.dim_x     + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*(z%para.dim_z);
 node_index[1] = (x+1)%para.dim_x + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*(z%para.dim_z);
 node_index[2] = x%para.dim_x     + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*(z%para.dim_z);
 node_index[3] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*(z%para.dim_z);
 node_index[4] = x%para.dim_x     + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*((z+1)%para.dim_z);
 node_index[5] = (x+1)%para.dim_x + para.dim_x*(y%para.dim_y)     + para.dim_x*para.dim_y*((z+1)%para.dim_z);
 node_index[6] = x%para.dim_x     + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
 node_index[7] = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);

 particle_force[part_index].f[0] = 0.f;
 particle_force[part_index].f[1] = 0.f;
 particle_force[part_index].f[2] = 0.f;

 interpolated_u1 = interpolated_u2 = interpolated_u3 = 0.f;
 #pragma unroll
 for(int i=0; i<8; ++i){
    float totmass=0.f;
    calc_m_from_n(n_a,node_index[i],mode);
    #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;ii++){
	totmass+=mode[0]+para.rho[ii]*para.agrid*para.agrid*para.agrid;
    } 
#ifndef SHANCHEN
    interpolated_u1 += (mode[1]/totmass)*delta[i];
    interpolated_u2 += (mode[2]/totmass)*delta[i];
    interpolated_u3 += (mode[3]/totmass)*delta[i];
#else //SHANCHEN
    interpolated_u1 += d_v[node_index[i]].v[0]/8.;  
    interpolated_u2 += d_v[node_index[i]].v[1]/8.;
    interpolated_u3 += d_v[node_index[i]].v[2]/8.;
#endif
 }

#ifdef SHANCHEN
 #pragma unroll
 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
  float solvation2 = particle_data[part_index].solvation[2*ii + 1];
   
  interpolated_rho[ii]  = 0.f;
  gradrho1 = gradrho2 = gradrho3 = 0.f;
  
 // TODO: should one introduce a density-dependent friction ?
  calc_mode(mode, n_a, node_index[0],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
  interpolated_rho[ii] += delta[0] * Rho; 
  partgrad1[ii*8 + 0] += Rho * solvation2;
  partgrad2[ii*8 + 0] += Rho * solvation2;
  partgrad3[ii*8 + 0] += Rho * solvation2;
  gradrho1 -=(delta[0] + delta[1]) * Rho; 
  gradrho2 -=(delta[0] + delta[2]) * Rho; 
  gradrho3 -=(delta[0] + delta[4]) * Rho; 

  calc_mode(mode, n_a, node_index[1],ii); 
  Rho = mode[0] +  para.rho[ii]*para.agrid*para.agrid*para.agrid; 
  interpolated_rho[ii] += delta[1] * Rho; 
  partgrad1[ii*8 + 1] -= Rho * solvation2;
  partgrad2[ii*8 + 1] += Rho * solvation2;
  partgrad3[ii*8 + 1] += Rho * solvation2;
  gradrho1 +=(delta[1] + delta[0]) * Rho; 
  gradrho2 -=(delta[1] + delta[3]) * Rho; 
  gradrho3 -=(delta[1] + delta[5]) * Rho; 
  
  calc_mode(mode, n_a, node_index[2],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[2] * Rho; 
  partgrad1[ii*8 + 2] += Rho * solvation2;
  partgrad2[ii*8 + 2] -= Rho * solvation2;
  partgrad3[ii*8 + 2] += Rho * solvation2;
  gradrho1 -=(delta[2] + delta[3]) * Rho; 
  gradrho2 +=(delta[2] + delta[0]) * Rho; 
  gradrho3 -=(delta[2] + delta[6]) * Rho; 

  calc_mode(mode, n_a, node_index[3],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[3] * Rho; 
  partgrad1[ii*8 + 3] -= Rho * solvation2;
  partgrad2[ii*8 + 3] -= Rho * solvation2;
  partgrad3[ii*8 + 3] += Rho * solvation2;
  gradrho1 +=(delta[3] + delta[2]) * Rho; 
  gradrho2 +=(delta[3] + delta[1]) * Rho; 
  gradrho3 -=(delta[3] + delta[7]) * Rho; 

  calc_mode(mode, n_a, node_index[4],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[4] * Rho; 
  partgrad1[ii*8 + 4] += Rho * solvation2;
  partgrad2[ii*8 + 4] += Rho * solvation2;
  partgrad3[ii*8 + 4] -= Rho * solvation2;
  gradrho1 -=(delta[4] + delta[5]) * Rho; 
  gradrho2 -=(delta[4] + delta[6]) * Rho; 
  gradrho3 +=(delta[4] + delta[0]) * Rho; 

  calc_mode(mode, n_a, node_index[5],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[5] * Rho; 
  partgrad1[ii*8 + 5] -= Rho * solvation2;
  partgrad2[ii*8 + 5] += Rho * solvation2;
  partgrad3[ii*8 + 5] -= Rho * solvation2;
  gradrho1 +=(delta[5] + delta[4]) * Rho; 
  gradrho2 -=(delta[5] + delta[7]) * Rho; 
  gradrho3 +=(delta[5] + delta[1]) * Rho; 

  calc_mode(mode, n_a, node_index[6],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[6] * Rho; 
  partgrad1[ii*8 + 6] += Rho * solvation2;
  partgrad2[ii*8 + 6] -= Rho * solvation2;
  partgrad3[ii*8 + 6] -= Rho * solvation2;
  gradrho1 -=(delta[6] + delta[7]) * Rho; 
  gradrho2 +=(delta[6] + delta[4]) * Rho; 
  gradrho3 +=(delta[6] + delta[2]) * Rho; 

  calc_mode(mode, n_a, node_index[7],ii);
  Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;	
  interpolated_rho[ii] += delta[7] * Rho; 
  partgrad1[ii*8 + 7] -= Rho * solvation2;
  partgrad2[ii*8 + 7] -= Rho * solvation2;
  partgrad3[ii*8 + 7] -= Rho * solvation2;
  gradrho1 +=(delta[7] + delta[6]) * Rho; 
  gradrho2 +=(delta[7] + delta[5]) * Rho; 
  gradrho3 +=(delta[7] + delta[3]) * Rho; 

  /* normalize the gradient to md units TODO: is that correct?*/
  gradrho1 *= para.agrid; 
  gradrho2 *= para.agrid; 
  gradrho3 *= para.agrid; 

  scforce[0+ii*3] += particle_data[part_index].solvation[2*ii] * gradrho1 ; 
  scforce[1+ii*3] += particle_data[part_index].solvation[2*ii] * gradrho2 ;
  scforce[2+ii*3] += particle_data[part_index].solvation[2*ii] * gradrho3 ;
  /* scforce is used also later...*/
  particle_force[part_index].f[0] += scforce[0+ii*3];
  particle_force[part_index].f[1] += scforce[1+ii*3];
  particle_force[part_index].f[2] += scforce[2+ii*3];
 }

#else // SHANCHEN is not defined
 /* for LB we do not reweight the friction force */
 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
	interpolated_rho[ii]=1.0;
 }

#endif // SHANCHEN

  /** calculate viscous force
   * take care to rescale velocities with time_step and transform to MD units
   * (Eq. (9) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */
 float rhotot=0;

 #pragma unroll
 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
	rhotot+=interpolated_rho[ii];
 }


 /* Viscous force */

 for(int ii=0; ii<LB_COMPONENTS; ++ii){ 
  viscforce[0+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(particle_data[part_index].v[0]/para.time_step - interpolated_u1*para.agrid/para.tau)/rhotot;
  viscforce[1+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(particle_data[part_index].v[1]/para.time_step - interpolated_u2*para.agrid/para.tau)/rhotot;
  viscforce[2+ii*3] -= interpolated_rho[ii]*para.friction[ii]*(particle_data[part_index].v[2]/para.time_step - interpolated_u3*para.agrid/para.tau)/rhotot;

#ifdef LB_ELECTROHYDRODYNAMICS
  viscforce[0+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[0]/rhotot;
  viscforce[1+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[1]/rhotot;
  viscforce[2+ii*3] += interpolated_rho[ii]*para.friction[ii] * particle_data[part_index].mu_E[2]/rhotot;
#endif

  /** add stochastic force of zero mean (Ahlrichs, Duenweg equ. 15)*/
#ifdef GAUSSRANDOM
  gaussian_random(rn_part);
  viscforce[0+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
  viscforce[1+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[1];
  gaussian_random(rn_part);
  viscforce[2+ii*3] += para.lb_coupl_pref2[ii]*rn_part->randomnr[0];
#else
  random_01(rn_part);
  viscforce[0+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[0]-0.5f);
  viscforce[1+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[1]-0.5f);
  random_01(rn_part);
  viscforce[2+ii*3] += para.lb_coupl_pref[ii]*(rn_part->randomnr[0]-0.5f);
#endif	  
  /** delta_j for transform momentum transfer to lattice units which is done in calc_node_force
  (Eq. (12) Ahlrichs and Duenweg, JCP 111(17):8225 (1999)) */

  particle_force[part_index].f[0] += viscforce[0+ii*3];
  particle_force[part_index].f[1] += viscforce[1+ii*3];
  particle_force[part_index].f[2] += viscforce[2+ii*3];
  /* the average force from the particle to surrounding nodes is transmitted back to preserve momentum */
  for(int node=0 ; node < 8 ; node++ ) { 
     particle_force[part_index].f[0] -= partgrad1[node+ii*8]/8.;
     particle_force[part_index].f[1] -= partgrad2[node+ii*8]/8.;
     particle_force[part_index].f[2] -= partgrad3[node+ii*8]/8.;
  }
  /* note that scforce is zero if SHANCHEN is not #defined */
  delta_j[0+3*ii] -= (scforce[0+ii*3]+viscforce[0+ii*3])*para.time_step*para.tau/para.agrid;
  delta_j[1+3*ii] -= (scforce[1+ii*3]+viscforce[1+ii*3])*para.time_step*para.tau/para.agrid;
  delta_j[2+3*ii] -= (scforce[2+ii*3]+viscforce[2+ii*3])*para.time_step*para.tau/para.agrid;  	
 }
}

/**calcutlation of the node force caused by the particles, with atomicadd due to avoiding race conditions 
	(Eq. (14) Ahlrichs and Duenweg, JCP 111(17):8225 (1999))
 * @param *delta		Pointer for the weighting of particle position (Input)
 * @param *delta_j		Pointer for the weighting of particle momentum (Input)
 * @param node_index		node index around (8) particle (Input)
 * @param node_f    		Pointer to the node force (Output).
*/
__device__ void calc_node_force(float *delta, float *delta_j, float * partgrad1, float * partgrad2, float * partgrad3,  unsigned int *node_index, LB_node_force_gpu node_f){
/* TODO: should the drag depend on the density?? */
/* NOTE: partgrad is not zero only if SHANCHEN is defined. It is initialized in calc_node_force. Alternatively one could 
         specialize this function to the single component LB */ 
 for(int ii=0; ii < LB_COMPONENTS; ++ii) { 
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[0+ii*3] + partgrad1[ii*8+0]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[1+ii*3] + partgrad2[ii*8+0]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[0]]), (delta[0]*delta_j[2+ii*3] + partgrad3[ii*8+0]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[0+ii*3] + partgrad1[ii*8+1]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[1+ii*3] + partgrad2[ii*8+1]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[1]]), (delta[1]*delta_j[2+ii*3] + partgrad3[ii*8+1]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[0+ii*3] + partgrad1[ii*8+2]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[1+ii*3] + partgrad2[ii*8+2]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[2]]), (delta[2]*delta_j[2+ii*3] + partgrad3[ii*8+2]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[0+ii*3] + partgrad1[ii*8+3]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[1+ii*3] + partgrad2[ii*8+3]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[3]]), (delta[3]*delta_j[2+ii*3] + partgrad3[ii*8+3]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[0+ii*3] + partgrad1[ii*8+4]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[1+ii*3] + partgrad2[ii*8+4]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[4]]), (delta[4]*delta_j[2+ii*3] + partgrad3[ii*8+4]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[0+ii*3] + partgrad1[ii*8+5]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[1+ii*3] + partgrad2[ii*8+5]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[5]]), (delta[5]*delta_j[2+ii*3] + partgrad3[ii*8+5]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[0+ii*3] + partgrad1[ii*8+6]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[1+ii*3] + partgrad2[ii*8+6]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[6]]), (delta[6]*delta_j[2+ii*3] + partgrad3[ii*8+6]));
                                                                                                    
  atomicadd(&(node_f.force[(0+ii*3)*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[0+ii*3] + partgrad1[ii*8+7]));
  atomicadd(&(node_f.force[(1+ii*3)*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[1+ii*3] + partgrad2[ii*8+7]));
  atomicadd(&(node_f.force[(2+ii*3)*para.number_of_nodes + node_index[7]]), (delta[7]*delta_j[2+ii*3] + partgrad3[ii*8+7]));
 }
}


/*********************************************************/
/** \name System setup and Kernel functions */
/*********************************************************/

/**kernel to calculate local populations from hydrodynamic fields given by the tcl values.
 * The mapping is given in terms of the equilibrium distribution.
 *
 * Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 * Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 * @param n_a		 Pointer to the lattice site (Input).
 * @param *gpu_check additional check if gpu kernel are executed(Input).
*/
__global__ void calc_n_equilibrium(LB_nodes_gpu n_a, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f) {
   /* TODO: this can handle only a uniform density, somehting similar, but local, 
            has to be called every time the fields are set by the user ! */ 
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  if(index<para.number_of_nodes){
       float mode[19*LB_COMPONENTS];
       #pragma unroll
       for(int ii=0;ii<LB_COMPONENTS;++ii) { 
     
         /** default values for fields in lattice units */
         float Rho = para.rho[ii]*para.agrid*para.agrid*para.agrid;
         float v[3] = { 0.0f, 0.0f, 0.0f };
         float pi[6] = { Rho*c_sound_sq, 0.0f, Rho*c_sound_sq, 0.0f, 0.0f, Rho*c_sound_sq };
     
         float rhoc_sq = Rho*c_sound_sq;
         float avg_rho = para.rho[ii]*para.agrid*para.agrid*para.agrid;
         float local_rho, local_j[3], *local_pi, trace;
     
         local_rho  = Rho;
     
         local_j[0] = Rho * v[0];
         local_j[1] = Rho * v[1];
         local_j[2] = Rho * v[2];
     
         local_pi = pi;
     
         /** reduce the pressure tensor to the part needed here. NOTE: this not true anymore for SHANCHEN if the densities are not uniform. FIXME*/
         local_pi[0] -= rhoc_sq;
         local_pi[2] -= rhoc_sq;
         local_pi[5] -= rhoc_sq;
     
         trace = local_pi[0] + local_pi[2] + local_pi[5];
     
         float rho_times_coeff;
         float tmp1,tmp2;
     
         /** update the q=0 sublattice */
         n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + index] = 1.f/3.f * (local_rho-avg_rho) - 1.f/2.f*trace;
     
         /** update the q=1 sublattice */
         rho_times_coeff = 1.f/18.f * (local_rho-avg_rho);
     
         n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
         n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
         n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
         n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
         n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;
         n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;
     
         /** update the q=2 sublattice */
         rho_times_coeff = 1.f/36.f * (local_rho-avg_rho);
     
         tmp1 = local_pi[0] + local_pi[2];
         tmp2 = 2.0f*local_pi[1];
         n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + index]  = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + index]  = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + index]  = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
         n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
     
         tmp1 = local_pi[0] + local_pi[5];
         tmp2 = 2.0f*local_pi[3];
     
         n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
         n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
     
         tmp1 = local_pi[2] + local_pi[5];
         tmp2 = 2.0f*local_pi[4];
     
         n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
         n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff + 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
         n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + index] = rho_times_coeff - 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
     
         /**set different seed for randomgen on every node */
         n_a.seed[index] = para.your_seed + index;
       }
       calc_m_from_n(n_a,index,mode);
       update_rho_v(mode,index,node_f,d_v);
  }
}

/** kernel to calculate local populations from hydrodynamic fields
 * from given flow field velocities.  The mapping is given in terms of
 * the equilibrium distribution.
 *
 * Eq. (2.15) Ladd, J. Fluid Mech. 271, 295-309 (1994)
 * Eq. (4) in Berk Usta, Ladd and Butler, JCP 122, 094902 (2005)
 *
 * @param n_a		   the current nodes array (double buffering!)
 * @param single_nodeindex the node to set the velocity for
 * @param velocity         the velocity to set
 */
__global__ void set_u_equilibrium(LB_nodes_gpu n_a, int single_nodeindex,float *velocity) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0){
  float v[3];
  float mode[4*LB_COMPONENTS];
  float rhoc_sq,avg_rho;
  float local_rho, local_j[3], *local_pi, trace;
  v[0] = velocity[0];
  v[1] = velocity[1];
  v[2] = velocity[2];
  #pragma unroll
  for(int ii=0;ii<LB_COMPONENTS;++ii) { 

    /** default values for fields in lattice units */
    calc_mode(&mode[4*ii], n_a, single_nodeindex,ii);
    float Rho = mode[0*4*ii] + para.rho[ii]*para.agrid*para.agrid*para.agrid; 

    float pi[6] = { Rho*c_sound_sq, 0.0f, Rho*c_sound_sq, 0.0f, 0.0f, Rho*c_sound_sq };

    rhoc_sq = Rho*c_sound_sq;
    avg_rho = para.rho[ii]*para.agrid*para.agrid*para.agrid;

    local_rho  = Rho;

    local_j[0] = Rho * v[0];
    local_j[1] = Rho * v[1];
    local_j[2] = Rho * v[2];


    local_pi = pi;

    /** reduce the pressure tensor to the part needed here. NOTE: this not true anymore for SHANCHEN if the densities are not uniform. FIXME*/
    /* there is much duplicated code from calc_n_equilibrium(). FIXME */
    local_pi[0] -= rhoc_sq; 
    local_pi[2] -= rhoc_sq;
    local_pi[5] -= rhoc_sq;

    trace = local_pi[0] + local_pi[2] + local_pi[5];

    float rho_times_coeff;
    float tmp1,tmp2;

    /** update the q=0 sublattice */
    n_a.vd[(0 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/3.f * (local_rho-avg_rho) - 1.f/2.f*trace;

    /** update the q=1 sublattice */
    rho_times_coeff = 1.f/18.f * (local_rho-avg_rho);

    n_a.vd[(1 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[(2 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/6.f*local_j[0] + 1.f/4.f*local_pi[0] - 1.f/12.f*trace;
    n_a.vd[(3 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[(4 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/6.f*local_j[1] + 1.f/4.f*local_pi[2] - 1.f/12.f*trace;
    n_a.vd[(5 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;
    n_a.vd[(6 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/6.f*local_j[2] + 1.f/4.f*local_pi[5] - 1.f/12.f*trace;

    /** update the q=2 sublattice */
    rho_times_coeff = 1.f/36.f * (local_rho-avg_rho);

    tmp1 = local_pi[0] + local_pi[2];
    tmp2 = 2.0f*local_pi[1];
    n_a.vd[(7 + ii*LBQ ) * para.number_of_nodes + single_nodeindex]  = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(8 + ii*LBQ ) * para.number_of_nodes + single_nodeindex]  = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[1]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(9 + ii*LBQ ) * para.number_of_nodes + single_nodeindex]  = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[1]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[0] + local_pi[5];
    tmp2 = 2.0f*local_pi[3];

    n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[0]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[0]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

    tmp1 = local_pi[2] + local_pi[5];
    tmp2 = 2.0f*local_pi[4];

    n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[1]+local_j[2]) + 1.f/8.f*(tmp1+tmp2) - 1.f/24.f*trace;
    n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff + 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;
    n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = rho_times_coeff - 1.f/12.f*(local_j[1]-local_j[2]) + 1.f/8.f*(tmp1-tmp2) - 1.f/24.f*trace;

  }
  }
}


/**calculate mass of the whole fluid kernel
 * @param *sum				Pointer to result storage value (Output)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void calc_mass(LB_nodes_gpu n_a, float *sum) {
  float mode[4];

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    for(int ii=0;ii<LB_COMPONENTS;++ii) { 
      calc_mode(mode, n_a, index,ii);
      float Rho = mode[0] + para.rho[ii]*para.agrid*para.agrid*para.agrid;
      atomicadd(&(sum[0]), Rho);
    }
  }
}

/** (re-)initialization of the node force / set up of external force in lb units
 * @param node_f		Pointer to local node force (Input)
*/
__global__ void reinit_node_force(LB_node_force_gpu node_f){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
   #pragma unroll
   for(int ii=0;ii<LB_COMPONENTS;++ii){
#ifdef EXTERNAL_FORCES
    if(para.external_force){
      node_f.force[(0+ii*3)*para.number_of_nodes + index] = para.ext_force[0]*para.agrid*para.agrid*para.agrid*para.agrid*para.tau*para.tau;
      node_f.force[(1+ii*3)*para.number_of_nodes + index] = para.ext_force[1]*para.agrid*para.agrid*para.agrid*para.agrid*para.tau*para.tau;
      node_f.force[(2+ii*3)*para.number_of_nodes + index] = para.ext_force[2]*para.agrid*para.agrid*para.agrid*para.agrid*para.tau*para.tau;
    }
    else{
      node_f.force[(0+ii*3)*para.number_of_nodes + index] = 0.0f;
      node_f.force[(1+ii*3)*para.number_of_nodes + index] = 0.0f;
      node_f.force[(2+ii*3)*para.number_of_nodes + index] = 0.0f;
    }
#else
    node_f.force[(0+ii*3)*para.number_of_nodes + index] = 0.0f;
    node_f.force[(1+ii*3)*para.number_of_nodes + index] = 0.0f;
    node_f.force[(2+ii*3)*para.number_of_nodes + index] = 0.0f;
#endif
   }
  }
}


/**set extern force on single nodes kernel
 * @param n_extern_nodeforces		number of nodes (Input)
 * @param *extern_nodeforces		Pointer to extern node force array (Input)
 * @param node_f			node force struct (Output)
*/
__global__ void init_extern_nodeforces(int n_extern_nodeforces, LB_extern_nodeforce_gpu *extern_nodeforces, LB_node_force_gpu node_f){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  float factor=powf(para.agrid,4)*para.tau*para.tau;
  if(index<n_extern_nodeforces){
   #pragma unroll
   for(int ii=0;ii<LB_COMPONENTS;++ii){
    node_f.force[(0+ii*3)*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[0] * factor;
    node_f.force[(1+ii*3)*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[1] * factor;
    node_f.force[(2+ii*3)*para.number_of_nodes + extern_nodeforces[index].index] = extern_nodeforces[index].force[2] * factor;
   }
  }
}

#ifdef SHANCHEN

/** 
 * @param single_nodeindex	Single node index        (Input)
 * @param *mode			Pointer to the local register values mode (Output)
 * @param n_a			Pointer to local node residing in array a(Input)
*/
__device__ __inline__ float calc_massmode(LB_nodes_gpu n_a, int single_nodeindex, int component_index){
	
  /** mass mode */
  float mode;
  mode = n_a.vd[(0 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(1 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(2 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] 
          + n_a.vd[(3 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(4 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(5 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
          + n_a.vd[(6 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(7 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(8 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
          + n_a.vd[(9 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(10 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(11 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(12 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
          + n_a.vd[(13 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(14 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(15 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(16 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex]
          + n_a.vd[(17 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex] + n_a.vd[(18 + component_index*LBQ ) * para.number_of_nodes + single_nodeindex];
 mode += para.rho[component_index]*para.agrid*para.agrid*para.agrid;

 return mode;
}


__device__ __inline__ void calc_shanchen_contribution(LB_nodes_gpu n_a,int component_index, int x, int y, int z, float *p){ 
      float tmp_p[3]={0.f,0.f,0.f};
      float pseudo;
      int index;
      index  = (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/18.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/18.f;

      index  = x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]+=pseudo/18.f;

      index  = x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]-=pseudo/18.f;

      index  = x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[2]+=pseudo/18.f;	

      index  = x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[2]-=pseudo/18.f;

      index  = (x+1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/36.f;
      tmp_p[1]+=pseudo/36.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/36.f;
      tmp_p[1]-=pseudo/36.f;

      index  = (x+1)%para.dim_x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/36.f;
      tmp_p[1]-=pseudo/36.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*z;
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/36.f;
      tmp_p[1]+=pseudo/36.f;

      index  = (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/36.f;
      tmp_p[2]+=pseudo/36.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/36.f;
      tmp_p[2]-=pseudo/36.f;

      index  = (x+1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]+=pseudo/36.f;
      tmp_p[2]-=pseudo/36.f;

      index  = (para.dim_x+x-1)%para.dim_x + para.dim_x*y + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[0]-=pseudo/36.f;
      tmp_p[2]+=pseudo/36.f;

      index  = x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]+=pseudo/36.f;
      tmp_p[2]+=pseudo/36.f;

      index  = x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]-=pseudo/36.f;
      tmp_p[2]-=pseudo/36.f;

      index  = x + para.dim_x*((y+1)%para.dim_y) + para.dim_x*para.dim_y*((para.dim_z+z-1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]+=pseudo/36.f;
      tmp_p[2]-=pseudo/36.f;

      index  = x + para.dim_x*((para.dim_y+y-1)%para.dim_y) + para.dim_x*para.dim_y*((z+1)%para.dim_z);
      pseudo =  calc_massmode(n_a,index,component_index);
      tmp_p[1]-=pseudo/36.f;
      tmp_p[2]+=pseudo/36.f;
  
      p[0]=tmp_p[0];
      p[1]=tmp_p[1];
      p[2]=tmp_p[2];
}

/** function to calc shanchen forces 
 * @param *mode			Pointer to the local register values mode (Output)
 * @param n_a			Pointer to local node residing in array a(Input)
 * @param node_f		Pointer to local node force (Input)
*/
__global__ void lb_shanchen_GPU(LB_nodes_gpu n_a,LB_node_force_gpu node_f){
#ifndef D3Q19
#error Lattices other than D3Q19 not supported
#endif
#if ( LB_COMPONENTS == 1  ) 
  #warning shanchen forces not implemented 
#else  
  
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int xyz[3];
  float pseudo;

  if(index<para.number_of_nodes){
     /*Let's first identify the neighboring nodes */
     index_to_xyz(index, xyz);
     int x = xyz[0];
     int y = xyz[1];
     int z = xyz[2];
     
     #pragma unroll
     for(int ii=0;ii<LB_COMPONENTS;ii++){ 
       float p[3]={0.f,0.f,0.f};
       pseudo =  calc_massmode(n_a,index,ii);
       #pragma unroll
       for(int jj=0;jj<LB_COMPONENTS;jj++){ 
             float tmpp[3]={0.f,0.f,0.f};
             calc_shanchen_contribution(n_a, jj, x,y,z, tmpp);
// FIXME  coupling HAS to be rescaled with agrid....
             p[0] += - para.coupling[(LB_COMPONENTS)*ii+jj]  * pseudo  * tmpp[0];
             p[1] += - para.coupling[(LB_COMPONENTS)*ii+jj]  * pseudo  * tmpp[1];
             p[2] += - para.coupling[(LB_COMPONENTS)*ii+jj]  * pseudo  * tmpp[2];
       }
       node_f.force[(0+ii*3)*para.number_of_nodes + index]+=p[0];
       node_f.force[(1+ii*3)*para.number_of_nodes + index]+=p[1];
       node_f.force[(2+ii*3)*para.number_of_nodes + index]+=p[2];
     }
  }
#endif 
  return; 
}

#endif //SHANCHEN

/** kernel to set the local density
 *
 * @param n_a		   the current nodes array (double buffering!)
 * @param single_nodeindex the node to set the velocity for
 * @param rho              the density to set
 */
__global__ void set_rho(LB_nodes_gpu n_a,  LB_rho_v_gpu *d_v, int single_nodeindex,float *rho) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  /*Note: this sets the velocities to zero */
  if(index == 0){
     float local_rho;
     #pragma unroll
     for(int ii=0;ii<LB_COMPONENTS;++ii) { 
       /** default values for fields in lattice units */
       local_rho = (rho[ii]-para.rho[ii])*para.agrid*para.agrid*para.agrid;
       d_v[single_nodeindex].rho[ii]=rho[ii];
       n_a.vd[(0  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/3.f * local_rho ;
       n_a.vd[(1  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(2  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(3  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(4  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(5  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(6  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/18.f * local_rho ;
       n_a.vd[(7  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(8  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(9  + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(10 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(11 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(12 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(13 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(14 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(15 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(16 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(17 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
       n_a.vd[(18 + ii*LBQ ) * para.number_of_nodes + single_nodeindex] = 1.f/36.f * local_rho ;
     }
  }
}

/**set the boundary flag for all boundary nodes
 * @param boundary_node_list    The indices of the boundary nodes
 * @param boundary_index_list   The flag representing the corresponding boundary
 * @param number_of_boundnodes	The number of boundary nodes
 * @param n_a			Pointer to local node residing in array a (Input)
 * @param n_b			Pointer to local node residing in array b (Input)
*/
__global__ void init_boundaries(int *boundary_node_list, int *boundary_index_list, int number_of_boundnodes, LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<number_of_boundnodes){
    n_a.boundary[boundary_node_list[index]] = boundary_index_list[index];
    n_b.boundary[boundary_node_list[index]] = boundary_index_list[index];
  }	
}

/**reset the boundary flag of every node
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param n_b		Pointer to local node residing in array b (Input)	
*/
__global__ void reset_boundaries(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  size_t index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    n_a.boundary[index] = n_b.boundary[index] = 0;
  }
}

/** integrationstep of the lb-fluid-solver
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param n_b		Pointer to local node residing in array b (Input)
 * @param *d_v		Pointer to local device values (Input)
 * @param node_f	Pointer to local node force (Input)
*/
__global__ void integrate(LB_nodes_gpu n_a, LB_nodes_gpu n_b, LB_rho_v_gpu *d_v, LB_node_force_gpu node_f){
  /**every node is connected to a thread via the index*/
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  /**the 19 moments (modes) are only temporary register values */
  float mode[19*LB_COMPONENTS];
  LB_randomnr_gpu rng;

  if( index < para.number_of_nodes ){
    /** storing the seed into a register value*/
    rng.seed = n_a.seed[index];
    /**calc_m_from_n*/
    calc_m_from_n(n_a, index, mode);
    /**lb_relax_modes*/
    relax_modes(mode, index, node_f,d_v);
    /**lb_thermalize_modes */
    if (para.fluct){thermalize_modes(mode, index, &rng);}
#if  defined(EXTERNAL_FORCES)  ||   defined (SHANCHEN)  
    /**if external force is used apply node force */
    apply_forces(index, mode, node_f,d_v);
#else
    /**if partcles are used apply node forces*/
    if (para.number_of_particles) apply_forces(index, mode, node_f,d_v); 
#endif
    /**lb_calc_n_from_modes_push*/
    normalize_modes(mode);
    /**calc of velocity densities and streaming with pbc*/
    calc_n_from_modes_push(n_b, mode, index);
    /** rewriting the seed back to the global memory*/
    n_b.seed[index] = rng.seed;
  }  
}
/** fill buffers for multi gpu code
 * @param n_c	    	Pointer to local node residing in array a (Input)
 * @param *buffer		Pointer to local buffer (Input)
*/
__global__ void write_buffer(LB_nodes_gpu n_c, float* buffer){

  /**every node is connected to a thread via the index*/
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  /**the 19 moments (modes) are only temporary register values */

  if(index<para.number_of_nodes){
    //store buffer values at thier destination in lb nodes struct
    write_n_from_buffer(n_c, buffer, index);
  }
}
/** init buffers for multi gpu code
 * @param *s_buf    Pointer to send buffer (Input)
 * @param *r_buf		Pointer to receive buffer (Input)
*/
__global__ void init_buf(float* s_buf, float* r_buf){

  /**every node is connected to a thread via the index*/
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
//if(index == 0)printf("imax %i\n", (5*2*(para.number_of_halo_nodes[0]+para.number_of_halo_nodes[1]+para.number_of_halo_nodes[2])));
  if(index<(5*2*(para.number_of_halo_nodes[0]+para.number_of_halo_nodes[1]+para.number_of_halo_nodes[2]))){
    //store buffer values at thier destination in lb nodes struct
    s_buf[index] = 0.0;
    r_buf[index] = 0.0;
  }
}
/** part interaction kernel
 * @param n_a				Pointer to local node residing in array a (Input)
 * @param *particle_data		Pointer to the particle position and velocity (Input)
 * @param *particle_force		Pointer to the particle force (Input)
 * @param *part				Pointer to the rn array of the particles (Input)
 * @param node_f			Pointer to local node force (Input)
*/
__global__ void calc_fluid_particle_ia(LB_nodes_gpu n_a, CUDA_particle_data *particle_data, CUDA_particle_force *particle_force, LB_node_force_gpu node_f, CUDA_particle_seed *part, LB_rho_v_gpu *d_v){
	
  unsigned int part_index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  unsigned int node_index[8];
  float delta[8];
  float delta_j[3*LB_COMPONENTS]; 
  float partgrad1[8*LB_COMPONENTS]; 
  float partgrad2[8*LB_COMPONENTS]; 
  float partgrad3[8*LB_COMPONENTS]; 
  LB_randomnr_gpu rng_part;
  if(part_index<para.number_of_particles){

    rng_part.seed = part[part_index].seed;
    /**force acting on the particle. delta_j will be used later to compute the force that acts back onto the fluid. */
    calc_viscous_force(n_a, delta, partgrad1, partgrad2, partgrad3, particle_data, particle_force, part_index, &rng_part, delta_j, node_index,d_v);
    calc_node_force(delta, delta_j, partgrad1, partgrad2, partgrad3, node_index, node_f); 
    /**force which acts back to the fluid node */
    part[part_index].seed = rng_part.seed;		
  }
}

#ifdef LB_BOUNDARIES_GPU
/**Bounce back boundary read kernel
 * @param n_a					Pointer to local node residing in array a (Input)
 * @param n_b					Pointer to local node residing in array b (Input)
 * @param LB_boundary_velocity 			The constant velocity at the boundary, set by the user (Input)
 * @param LB_boundary_force 			The force on the boundary nodes (Output)
*/
__global__ void bb_read(LB_nodes_gpu n_a, LB_nodes_gpu n_b, float* LB_boundary_velocity, float* LB_boundary_force){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    bounce_back_read(n_b, n_a, index, LB_boundary_velocity, LB_boundary_force);
  }
}

/**Bounce back boundary write kernel
 * @param n_a					Pointer to local node residing in array a (Input)
 * @param n_b					Pointer to local node residing in array b (Input)
*/
__global__ void bb_write(LB_nodes_gpu n_a, LB_nodes_gpu n_b){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    bounce_back_write(n_b, n_a, index);
  }
}
/**Bounce back boundary write kernel
 * @param n_a					Pointer to local node residing in array a (Input)
 * @param n_b					Pointer to local node residing in array b (Input)
*/
__global__ void bb_write_buffer(LB_nodes_gpu n_a, LB_nodes_gpu n_b, float* buffer){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  float mode[19];

  if(index<para.number_of_nodes){
    //TODO write a single function to fill buffer directly
    calc_m_from_n(n_b, index, mode);
    normalize_modes(mode);
    calc_n_from_modes_buffer(n_b, buffer, mode, index);
  }
}
#endif

/** get physical values of the nodes (density, velocity, ...)
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param *p_v		Pointer to local print values (Output)
 * @param *d_v		Pointer to local device values (Input)
*/
__global__ void get_mesoscopic_values_in_MD_units(LB_nodes_gpu n_a, LB_rho_v_pi_gpu *p_v,LB_rho_v_gpu *d_v) {
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index < para.number_of_nodes) {
    float mode[19*LB_COMPONENTS];
    calc_m_from_n(n_a, index, mode);
    calc_values_in_MD_units(n_a, mode, p_v, d_v, index, index);
  }
}
/** get physical values of the nodes without halonodes (density, velocity, ...)
 * @param n_a		Pointer to local node residing in array a (Input)
 * @param *p_v		Pointer to local device values (Input)
*/
__global__ void get_mesoscopic_values_in_MD_units_wo_halo(LB_nodes_gpu n_a, LB_rho_v_pi_gpu *p_v,LB_rho_v_gpu *d_v) {
 
  //TODO remove single_node?
  unsigned int singlenode = 0;
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
    float mode[4];
    calc_mode(mode, n_a, index);
    //TODO rename/adjust following function
    //calc_values_wo_halo(n_a, mode, d_v, index, singlenode, node_f);
    calc_values_in_MD_units(n_a, mode, p_v, d_v, index, index);
  }
}
/** get boundary flags
 *  @param n_a	              Pointer to local node residing in array a (Input)
 *  @param device_bound_array Pointer to local device values (Input)
 */
__global__ void lb_get_boundaries(LB_nodes_gpu n_a, unsigned int *device_bound_array){

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index<para.number_of_nodes){
   device_bound_array[index] = n_a.boundary[index];
  }
}



/**print single node values kernel
 * @param single_nodeindex		index of the node (Input)
 * @param *d_p_v			Pointer to result storage array (Input)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void lb_print_node(int single_nodeindex, LB_rho_v_pi_gpu *d_p_v, LB_nodes_gpu n_a, LB_rho_v_gpu * d_v){
	
  float mode[19*LB_COMPONENTS];
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0) {
     calc_m_from_n(n_a, single_nodeindex, mode);
     
     /* the following actually copies rho and v from d_v, and calculates pi */
     calc_values_in_MD_units(n_a, mode, d_p_v, d_v, single_nodeindex, 0);
  }
}
__global__ void momentum(LB_nodes_gpu n_a, LB_rho_v_gpu * d_v, LB_node_force_gpu node_f, float *sum) {

  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;
  if(index<para.number_of_nodes){
    float j[3]={0.f,0.f,0.f};
    float mode[4];
    for(int ii=0 ; ii < LB_COMPONENTS ; ii++ ) { 
        calc_mode(mode, n_a, index,ii);
        j[0] += mode[1]+node_f.force[(0+ii*3)*para.number_of_nodes + index];
        j[1] += mode[2]+node_f.force[(1+ii*3)*para.number_of_nodes + index];
        j[2] += mode[3]+node_f.force[(2+ii*3)*para.number_of_nodes + index];
    }
#ifdef LB_BOUNDARIES_GPU
    if(n_a.boundary[index]){
	j[0]=j[1]=j[2]=0.0f;
    }
#endif
    atomicadd(&(sum[0]), j[0]); 
    atomicadd(&(sum[1]), j[1]); 
    atomicadd(&(sum[2]), j[2]); 
  }

}

/**print single node boundary flag
 * @param single_nodeindex		index of the node (Input)
 * @param *device_flag			Pointer to result storage array (Input)
 * @param n_a				Pointer to local node residing in array a (Input)
*/
__global__ void lb_get_boundary_flag(int single_nodeindex, unsigned int *device_flag, LB_nodes_gpu n_a){
	
  unsigned int index = blockIdx.y * gridDim.x * blockDim.x + blockDim.x * blockIdx.x + threadIdx.x;

  if(index == 0){
    device_flag[0] = n_a.boundary[single_nodeindex];
  }	
}

/**********************************************************************/
/* Host functions to setup and call kernels*/
/**********************************************************************/

void lb_get_para_pointer(LB_parameters_gpu** pointeradress) {
  if(hipGetSymbolAddress((void**) pointeradress, para) != hipSuccess) {
    printf("Trouble getting address of LB parameters.\n"); //TODO give proper error message
    exit(1);
  }
}

void lb_get_lbpar_pointer(LB_parameters_gpu** pointeradress) {
  *pointeradress = &lbpar_gpu;
}
/**get hardware info of GPUs
 * @param lbpar_gpu.number_of_gpus
*/
void hw_get_dev_count(){
  
  cuda_check_errors(hipGetDeviceCount(&lbdevicepar_gpu.number_of_gpus));

}
/**get hardware info of GPUs
 * @param dev device number
*/
void hw_set_dev(int dev){

  cuda_check_errors(hipSetDevice(dev)); 
  //printf("host no. %i set gpu no. %i \n", this_node, dev);

}
/**get hardware info of GPUs
 * @param dev device number
*/
int lb_set_devices(int* dev, int count){

  lbdevicepar_gpu.number_of_gpus = count;
  //printf("number of GPUs %i \n", count);
  return ES_OK;
}

/**get hardware info of GPUs
 * @param dev device number
*/
int lb_get_devices(int* dev){

  int count;
  count = lbdevicepar_gpu.number_of_gpus;
  //printf("number of GPUs %i \n", count);
  return count;
}
void lb_reinit_plan(){

  LB_TRACE(printf("node %i reinit_plan: number of gpus %i\n", this_node, lbdevicepar_gpu.number_of_gpus));
//only one gpu per cpu node so far!!!
  lbdevicepar_gpu.gpus_per_cpu = 1;
  if(plan_initflag == 1){
    free(plan);
  }
  //lbpar_gpu.number_of_gpus = count;
//check if number of nodes suits to number of gpus
  if(lbdevicepar_gpu.number_of_gpus == 1){
    if(this_node == 0){
      lbdevicepar_gpu.gpu_number = lbdevicepar_gpu.devices[0];
      //malloc plan struct for each gpu per cpu node 
      plan = (plan_gpu*)malloc(lbdevicepar_gpu.gpus_per_cpu*sizeof(plan_gpu));
      gpu_n = lbdevicepar_gpu.gpus_per_cpu;
      //printf("thisnode %i gpun %i\n",this_node, gpu_n);
      for(int g = 0; g < gpu_n; ++g){
        plan[g].initflag = 0;
      }
    }
  }else{
    if (n_nodes%lbdevicepar_gpu.number_of_gpus == 1 || lbdevicepar_gpu.number_of_gpus%n_nodes == 1){
      printf("ERROR: Number of MPI process must be multiple of number of GPUs!!!\n");
      exit(-1);
    }
//   so far one needs at least #n mpi process to use #n gpus
//  distribute gpu to cpu nodes  
    //lbdevicepar_gpu.gpu_number = this_node%lbpar_gpu.number_of_gpus;
    //printf("thisnode %i devs %i %i\n",this_node, lbpar_gpu.devices[0], lbpar_gpu.devices[1]);
    lbdevicepar_gpu.gpu_number = lbdevicepar_gpu.devices[this_node%lbdevicepar_gpu.number_of_gpus];
    printf("thisnode %i gpu_number %i\n",this_node, lbdevicepar_gpu.gpu_number);
    //printf("par gpu dimx address %p \n", &lbpar_gpu.dim_x);
    //printf("GPU number: %i -> this_node %i\n", lbdevicepar_gpu.gpu_number, this_node);
    hw_set_dev(lbdevicepar_gpu.gpu_number);
    //malloc plan struct for each gpu per cpu node 
    plan = (plan_gpu*)malloc(lbdevicepar_gpu.gpus_per_cpu*sizeof(plan_gpu));
    gpu_n = lbdevicepar_gpu.gpus_per_cpu;
    plan_initflag = 1;
    //printf("thisnode %i gpun %i\n",this_node, gpu_n);
    for(int g = 0; g < gpu_n; ++g){
      plan[g].initflag = 0;
    }
  }
}

void lb_setup_plan(){

  LB_TRACE(printf("node %i setup_plan gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
//only one gpu per cpu node so far!!!
  lbdevicepar_gpu.gpus_per_cpu = 1;
  hw_get_dev_count();
//check if number of nodes suits to number of gpus
  if (n_nodes%lbdevicepar_gpu.number_of_gpus == 1 || lbdevicepar_gpu.number_of_gpus%n_nodes == 1){
    printf("ERROR: Number of MPI process must be multiple of number of GPUs!!!\n");
    exit(-1);
  }
// so far one needs at least #n mpi process to use #n gpus
//distribute gpu to cpu nodes  
  for(int i = 0; i<n_nodes; ++i){
    /* decided which CPU nodes belongs to which GPU*/
    if (this_node == i) {
      lbdevicepar_gpu.gpu_number = this_node%lbdevicepar_gpu.number_of_gpus;
      //printf("par gpu dimx address %p \n", &lbpar_gpu.dim_x);
      //printf("GPU number: %i -> this_node %i\n", lbpar_gpu.gpu_number, this_node);
      hw_set_dev(lbdevicepar_gpu.gpu_number);
    }
  }
  //malloc plan struct for each gpu per cpu node 
  plan = (plan_gpu*)malloc(lbdevicepar_gpu.gpus_per_cpu*sizeof(plan_gpu));
  gpu_n = lbdevicepar_gpu.gpus_per_cpu;
  plan_initflag=1;
  //printf("thisnode %i gpun %i\n",this_node, gpu_n);
  for(int g = 0; g < gpu_n; ++g){
    plan[g].initflag = 0;
  }
  /**communication for the multi gpu fluid called from host
 * @param *s_buf_h	Pointer to source host buffer
 * @param *r_buf_h	Pointer to receive host buffer
 * @param *s_buf_d	Pointer to source device buffer
 * @param *r_buf_d	Pointer to receive device buffer
 * @param buf_size	buffer size
 * @param sn	      send node
 * @param rn      	receive node
*/
int cuda_comm_p2p_indirect_MPI(float *s_buf_h, float *r_buf_h, float *s_buf_d, float *r_buf_d, int buf_size, int sn, int rn){

  //slowest but "always" available p2p copy
  MPI_Status status;
  // send node: copy of data from device to host and send it via MPI
#if 1
    hipMemcpy(s_buf_h, s_buf_d, buf_size*sizeof(float), hipMemcpyDeviceToHost);
    //for(int i=0; i<buf_size; ++i)
    //  printf("thisnode %i s_buf_h[%i]: %f \n", this_node, i, s_buf_h[i]);
#endif
    //sn: node which is send TO! and rn: node FROM which is received
    int error_code;
    error_code = MPI_Sendrecv(s_buf_h, buf_size, MPI_FLOAT, sn, 101, r_buf_h, buf_size, MPI_FLOAT, rn, 101,
                   MPI_COMM_WORLD, &status);
#if 1 
    //for(int i=0; i<buf_size; ++i)
    //  printf("thisnode %i r_buf_h[%i]: %f \n", this_node, i, r_buf_h[i]);
    hipMemcpy(r_buf_d, r_buf_h, buf_size*sizeof(float), hipMemcpyHostToDevice);
    //if any error ocours
#endif
    if (error_code != MPI_SUCCESS) {
      char error_string[BUFSIZ];
      int length_of_error_string, error_class;
      MPI_Error_class(error_code, &error_class);
      MPI_Error_string(error_class, error_string, &length_of_error_string);
      fprintf(stderr, "%3d: %s\n", this_node, error_string);
      MPI_Error_string(error_code, error_string, &length_of_error_string);
      fprintf(stderr, "%3d: %s\n", this_node, error_string);
      MPI_Abort(MPI_COMM_WORLD, error_code);
    }
  return 1;
}

/**send and receive the buffers for multi-GPU usage
 * @param s_buf_d pointer to send buffer of buffer IN the GPU memory
 * @param r_buf_d pointer to receive buffer of buffer IN the GPU memory
  */
int lb_send_recv_buffer(float* s_buf_d, float* r_buf_d){

  LB_TRACE(printf("node %i sebd_recv_buffer gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //empty cpu buffers for communcation betwenn 2 gpus via cpu mem
  float *s_buf_h, *r_buf_h;
  int send_node, recv_node;
  unsigned offset;
  size_t buffer_size = 2*(size_of_buffer[0] + size_of_buffer[1] + size_of_buffer[2]);
  unsigned count[3] = {5*lbpar_gpu.number_of_halo_nodes[0], 5*lbpar_gpu.number_of_halo_nodes[1], 5*lbpar_gpu.number_of_halo_nodes[2]};
  s_buf_h = (float*)malloc(buffer_size);   
  r_buf_h = (float*)malloc(buffer_size);   
  //hipHostMalloc((void**)&s_buf_h, buffer_size);
  //hipHostMalloc((void**)&r_buf_h, buffer_size);
  //hipHostAlloc((void**)&s_buf_h, buffer_size, hipHostMallocMapped);
  //hipHostAlloc((void**)&r_buf_h, buffer_size, hipHostMallocMapped);
  //TODO  cuda_check_errors(hipHostAlloc((void**)&plan[g].send_buffer_d, 6*sizeof(float*), hipHostMallocMapped));   
  //    printf("thisnode %i node_grid: %i %i %i \n", this_node, node_grid[0], node_grid[1], node_grid[2]);
  /* send to right, recv from left i = 1, 7, 9, 11, 13 */
  send_node = node_neighbors[1];
  recv_node = node_neighbors[0];
  if (node_grid[0] > 1) {
    cuda_comm_p2p_indirect_MPI(s_buf_h, r_buf_h, s_buf_d, r_buf_d, count[0], send_node, recv_node);
    //printf("thisnode %i, :send_node: %i, recv_node: %i r_buf_h[0] %f\n",this_node, send_node, recv_node, r_buf_h[0]);
  } else {
    hipMemcpy(r_buf_d,s_buf_d,size_of_buffer[0], hipMemcpyDeviceToDevice);
   }
  /* send to left, recv from right i = 2, 8, 10, 12, 14 */
  send_node = node_neighbors[0];
  recv_node = node_neighbors[1];
    
  offset = 5*lbpar_gpu.number_of_halo_nodes[0];
  //printf("thisnode %i, offset %i, size_of_buffer[0] %i\n",this_node, offset, size_of_buffer[0]);
  if (node_grid[0] > 1) {
    cuda_comm_p2p_indirect_MPI((s_buf_h+offset), (r_buf_h+offset), (s_buf_d+offset), (r_buf_d+offset), count[0], send_node, recv_node);
  //printf("thisnode %i, :send_node: %i, recv_node: %i r_buf_h[0+offset] %f\n",this_node, send_node, recv_node, r_buf_h[0]);
  } else {
    hipMemcpy((r_buf_d+offset),(s_buf_d+offset),size_of_buffer[0], hipMemcpyDeviceToDevice);
   }
  /* send to front, recv from back i = 3, 7, 10, 15, 17 */
  send_node = node_neighbors[3];
  recv_node = node_neighbors[2];

  offset = 2*5*lbpar_gpu.number_of_halo_nodes[0];
  if (node_grid[1] > 1) {
    cuda_comm_p2p_indirect_MPI((s_buf_h+offset), (r_buf_h+offset), (s_buf_d+offset), (r_buf_d+offset), count[1], send_node, recv_node);
  } else {
    hipMemcpy((r_buf_d+offset),(s_buf_d+offset),size_of_buffer[1], hipMemcpyDeviceToDevice);
   }
  /* send to back, recv from front i = 4, 8, 9, 16, 18 */
  send_node = node_neighbors[2];
  recv_node = node_neighbors[3];
    
  offset = 5*(2*lbpar_gpu.number_of_halo_nodes[0] + lbpar_gpu.number_of_halo_nodes[1]);
  if (node_grid[1] > 1) {
    cuda_comm_p2p_indirect_MPI((s_buf_h+offset), (r_buf_h+offset), (s_buf_d+offset), (r_buf_d+offset), count[1], send_node, recv_node);
  } else {
    hipMemcpy((r_buf_d+offset),(s_buf_d+offset),size_of_buffer[1], hipMemcpyDeviceToDevice);
   }
  /* send to top, recv from bottom i = 5, 11, 14, 15, 18 */
  send_node = node_neighbors[5];
  recv_node = node_neighbors[4];
    
  offset = 5*2*(lbpar_gpu.number_of_halo_nodes[0] + lbpar_gpu.number_of_halo_nodes[1]);
  if (node_grid[2] > 1) {
    cuda_comm_p2p_indirect_MPI((s_buf_h+offset), (r_buf_h+offset), (s_buf_d+offset), (r_buf_d+offset), count[2], send_node, recv_node);
  } else {
    hipMemcpy((r_buf_d+offset),(s_buf_d+offset),size_of_buffer[2], hipMemcpyDeviceToDevice);
   }
  /* send to bottom, recv from top i = 6, 12, 13, 16, 17 */
  send_node = node_neighbors[4];
  recv_node = node_neighbors[5];
    
  offset = 5*2*(lbpar_gpu.number_of_halo_nodes[0] + lbpar_gpu.number_of_halo_nodes[1]) + 5*lbpar_gpu.number_of_halo_nodes[2];
  if (node_grid[2] > 1) {
    cuda_comm_p2p_indirect_MPI((s_buf_h+offset), (r_buf_h+offset), (s_buf_d+offset), (r_buf_d+offset), count[2], send_node, recv_node);
  } else {
    hipMemcpy((r_buf_d+offset),(s_buf_d+offset),size_of_buffer[2], hipMemcpyDeviceToDevice);
   }

  //printf("send_node: %i, recv_node: %i comm finished\n", send_node, recv_node);
  //printf("send_node: %i, recv_node: %i r_buf_h[0] %f\n", send_node, recv_node, r_buf_h[0]);
  lb_cp_buffer_in_vd();
  free(s_buf_h);
  free(r_buf_h);

  return 1;
}

/**copy of the velocity densities from buffer into vd array
 * @param 
*/
void lb_cp_buffer_in_vd(){

  LB_TRACE(printf("node %i cp_buffer_in_vd gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
  int g = 0; 
  //Attention GPU pointers
   // printf("node %i current pointer %p buffer %p\n", this_node, plan[g].current_nodes, plan[g].recv_buffer_d);
  KERNELCALL(write_buffer, dim_grid, threads_per_block, (*plan[g].current_nodes, plan[g].recv_buffer_d));

}
/**initialization for the lb gpu fluid called from host
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
*/
void lb_init_GPU(LB_parameters_gpu *lbpar_gpu){
#define free_and_realloc(var,size)\
  { if( (var) != NULL ) hipFree((var)); cuda_safe_mem(hipMalloc((void**)&var, size)); } 

  LB_TRACE(printf("node %i init_GPU gpu %i\n", this_node, lbdevicepar_gpu->gpu_number));
  LB_TRACE(printf("this_node: %i  local_box_l: %lf, %lf, %lf \n", this_node, local_box_l[0], local_box_l[1], local_box_l[2]));
  if (lbdevicepar_gpu->number_of_gpus == 1) {
    //dims stay like they are, just calc number of nodes 
    lbpar_gpu->number_of_nodes = (unsigned)(lbpar_gpu->dim_x*lbpar_gpu->dim_y*lbpar_gpu->dim_z);
    printf("Using only one GPU");
  }else{
    lbpar_gpu->dim_x = (unsigned)floor(local_box_l[0]/lbpar_gpu->agrid);
    lbpar_gpu->dim_y = (unsigned)floor(local_box_l[1]/lbpar_gpu->agrid);
    lbpar_gpu->dim_z = (unsigned)floor(local_box_l[2]/lbpar_gpu->agrid);
    lbpar_gpu->number_of_nodes_wo_halo = (unsigned) (lbpar_gpu->dim_x*lbpar_gpu->dim_y*lbpar_gpu->dim_z);
    //with halo in all three directions
    lbpar_gpu->dim_x += 2;
    lbpar_gpu->dim_y += 2;
    lbpar_gpu->dim_z += 2;
    //printf("dims: %u, %u, %u agrid %f\n", lbpar_gpu->dim_x, lbpar_gpu->dim_y, lbpar_gpu->dim_z, lbpar_gpu->agrid);
    lbpar_gpu->number_of_nodes = (unsigned) (lbpar_gpu->dim_x*lbpar_gpu->dim_y*lbpar_gpu->dim_z);
    //printf("init gpu number_of_nodes %i \n", lbpar_gpu->number_of_nodes);
    lbpar_gpu->number_of_halo_nodes[0] = (lbpar_gpu->dim_y*lbpar_gpu->dim_z);
    lbpar_gpu->number_of_halo_nodes[1] = (lbpar_gpu->dim_x*lbpar_gpu->dim_z);
    lbpar_gpu->number_of_halo_nodes[2] = (lbpar_gpu->dim_x*lbpar_gpu->dim_y);
    //printf("numberof_halonodes %i %i %i\n", lbpar_gpu->number_of_halo_nodes[0], lbpar_gpu->number_of_halo_nodes[1], lbpar_gpu->number_of_halo_nodes[2]);
  //
  }
  /** Allocate structs in device memory*/
  size_of_nodes_gpu = lbpar_gpu->number_of_nodes * 19 * sizeof(float);
  size_of_uint = lbpar_gpu->number_of_nodes * sizeof(unsigned int);
  size_of_3floats = lbpar_gpu->number_of_nodes * 3 * sizeof(float);
  stream = (hipStream_t*)malloc(gpu_n*sizeof(hipStream_t));
  size_of_rho_v     = lbpar_gpu->number_of_nodes * sizeof(LB_rho_v_gpu);
  size_of_rho_v_wo_halo     = lbpar_gpu->number_of_nodes_wo_halo * sizeof(LB_rho_v_gpu);
  size_of_rho_v_pi  = lbpar_gpu->number_of_nodes * sizeof(LB_rho_v_pi_gpu);
  size_of_rho_v_pi_wo_halo  = lbpar_gpu->number_of_nodes_wo_halo * sizeof(LB_rho_v_pi_gpu);

  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu->gpu_number));

    /** Allocate structs in device memory*/
    if(extended_values_flag==0) { 
                free_and_realloc(device_rho_v, size_of_rho_v);
    } else { 
              /* see the notes to the stucture device_rho_v_pi above...*/
                free_and_realloc(device_rho_v_pi, size_of_rho_v_pi);
    }

//FIXME check if this is needed and/or works
    //cuda_check_errors(hipDeviceReset());
    //cuda_check_errors(hipSetDeviceFlags(hipDeviceMapHost));
  /* TODO: this is a almost a copy copy of  device_rho_v thik about eliminating it, and maybe pi can be added to device_rho_v in this case*/
    free_and_realloc(plan[g].print_rho_v_pi  , size_of_rho_v_pi);
    free_and_realloc(plan[g].nodes_a.vd      , lbpar_gpu->number_of_nodes * 19 * LB_COMPONENTS * sizeof(float));
    free_and_realloc(plan[g].nodes_b.vd      , lbpar_gpu->number_of_nodes * 19 * LB_COMPONENTS * sizeof(float));   
    free_and_realloc(plan[g].node_f.force    , lbpar_gpu->number_of_nodes * 3  * LB_COMPONENTS * sizeof(float));

    free_and_realloc(plan[g].nodes_a.seed    , lbpar_gpu->number_of_nodes * sizeof( unsigned int));
    free_and_realloc(plan[g].nodes_a.boundary, lbpar_gpu->number_of_nodes * sizeof( unsigned int));
    free_and_realloc(plan[g].nodes_b.seed    , lbpar_gpu->number_of_nodes * sizeof( unsigned int));
    free_and_realloc(plan[g].nodes_b.boundary, lbpar_gpu->number_of_nodes * sizeof( unsigned int));


    /**write parameters in const memory*/
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
    cuda_check_errors(hipMemcpyToSymbol(HIP_SYMBOL(devpara), lbdevicepar_gpu, sizeof(LB_gpus)));
    //set up stream for execution order of gpu kernel functions
    hipStreamCreate(&stream[g]);

    if (lbdevicepar_gpu->number_of_gpus > 1) {
      //size of different buffers
      size_of_buffer[0] = 5 * lbpar_gpu->number_of_halo_nodes[0] * sizeof(float); 
      size_of_buffer[1] = 5 * lbpar_gpu->number_of_halo_nodes[1] * sizeof(float); 
      size_of_buffer[2] = 5 * lbpar_gpu->number_of_halo_nodes[2] * sizeof(float); 
    //printf("number of halo nodes %i %i %i\n", lbpar_gpu->number_of_halo_nodes[0],lbpar_gpu->number_of_halo_nodes[1],lbpar_gpu->number_of_halo_nodes[2]);

    //allocate buffer in GPU mem
      cuda_check_errors(hipMalloc((void**)&plan[g].send_buffer_d, 2*(size_of_buffer[0] + size_of_buffer[1] + size_of_buffer[2])));
      cuda_check_errors(hipMalloc((void**)&plan[g].recv_buffer_d, 2*(size_of_buffer[0] + size_of_buffer[1] + size_of_buffer[2])));
      //new thread and block dims to ensure that enough threads are executed to init complete buffer
      int threads_per_block_b = 64;
      int blocks_per_grid_y_b = 4;
      int blocks_per_grid_x_b = ((5*2*(lbpar_gpu->number_of_halo_nodes[0]+lbpar_gpu->number_of_halo_nodes[1]+lbpar_gpu->number_of_halo_nodes[2])) + threads_per_block_b * blocks_per_grid_y_b - 1) /(threads_per_block_b * blocks_per_grid_y_b);
      dim3 dim_grid_b = make_uint3(blocks_per_grid_x_b, blocks_per_grid_y_b, 1);

      KERNELCALL(init_buf, dim_grid_b, threads_per_block_b, (plan[g].send_buffer_d, plan[g].recv_buffer_d));

    }
    //set flag to one for release of gpu memory 
    plan[g].initflag = 1;
  
    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

    //should not be needed anymore
#if 0
  /** values for the particle kernel */
  int threads_per_block_particles = 64;
  int blocks_per_grid_particles_y = 4;
  int blocks_per_grid_particles_x = (lbpar_gpu->number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
  dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);
#endif   

  #ifdef SHANCHEN
  // TODO FIXME: 
  /* We must add shan-chen forces, which are zero only if the densities are uniform*/
  #endif

  /** calc of veloctiydensities from given parameters and initialize the Node_Force array with zero */
    KERNELCALL(calc_n_equilibrium, dim_grid, threads_per_block, (plan[g].nodes_a, plan[g].device_rho_v ,plan[g].node_f));	
    KERNELCALL(reinit_node_force, dim_grid, threads_per_block, (plan[g].node_f));
    KERNELCALL(reset_boundaries, dim_grid, threads_per_block, (plan[g].nodes_a, plan[g].nodes_b));
  

    //set a nodes as current nodes due to equilibration values stored in there
    plan[g].current_nodes = &plan[g].nodes_a;
    //and use them in the first integration loop
    plan[g].intflag = 1;
    //printf("current pointer %p\n", plan[g].current_nodes->vd);
    //printf("init current pointer %p nodes a %p\n", plan[g].current_nodes, &plan[g].nodes_a);
    //printf("init send_buf %p recv_buf %p\n", plan[g].send_buffer_d, plan[g].recv_buffer_d);
    ///barrier for alle init kernels 
    //hipStreamSynchronize(stream[g]);
    cuda_check_errors(hipDeviceSynchronize());
    //hipDeviceSynchronize();
  }
}
/** reinitialization for the lb gpu fluid called from host
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
*/
void lb_reinit_GPU(LB_parameters_gpu *lbpar_gpu){
//FIXME
  LB_TRACE(printf("node %i reinit_GPU gpu %i\n", this_node, lbdevicepar_gpu->gpu_number));
  //begin loop over devices i
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu->gpu_number));
    /**write parameters in const memory*/
    cuda_check_errors(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
    cuda_check_errors(hipMemcpyToSymbol(HIP_SYMBOL(devpara), lbdevicepar_gpu, sizeof(LB_gpus)));
  
    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

    /** calc of veloctiydensities from given parameters and initialize the Node_Force array with zero */
    KERNELCALL(calc_n_equilibrium, dim_grid, threads_per_block, (plan[g].nodes_a, plan[g].device_rho_v, plan[g].node_f));
  }
}

/**setup and call particle reallocation from the host
 * @param *lbpar_gpu	Pointer to parameters to setup the lb field
*/
void lb_realloc_particle_GPU_leftovers(LB_parameters_gpu *lbpar_gpu){

  //copy parameters, especially number of parts to gpu mem
  cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
}

#ifdef LB_BOUNDARIES_GPU
/** setup and call boundaries from the host
 * @param host_n_lb_boundaries number of LB boundaries
 * @param number_of_boundnodes	number of boundnodes
 * @param host_boundary_node_list    The indices of the boundary nodes
 * @param host_boundary_index_list   The flag representing the corresponding boundary
 * @param host_LB_Boundary_velocity 			The constant velocity at the boundary, set by the user (Input)
*/
void lb_init_boundaries_GPU(int host_n_lb_boundaries, int number_of_boundnodes, int *host_boundary_node_list, int* host_boundary_index_list, float* host_LB_Boundary_velocity){
  LB_TRACE(printf("node %i init_boundaries_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices i
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    int temp = host_n_lb_boundaries;

    size_of_boundindex = number_of_boundnodes*sizeof(int);
    cuda_safe_mem(hipMalloc((void**)&boundary_node_list, size_of_boundindex));
    cuda_safe_mem(hipMalloc((void**)&boundary_index_list, size_of_boundindex));
    cuda_safe_mem(hipMemcpy(boundary_index_list, host_boundary_index_list, size_of_boundindex, hipMemcpyHostToDevice));
    cuda_safe_mem(hipMemcpy(boundary_node_list, host_boundary_node_list, size_of_boundindex, hipMemcpyHostToDevice));

    cuda_safe_mem(hipMalloc((void**)&plan[g].lb_boundary_force   , 3*host_n_lb_boundaries*sizeof(float)));
    cuda_safe_mem(hipMalloc((void**)&plan[g].lb_boundary_velocity, 3*host_n_lb_boundaries*sizeof(float)));
    cuda_safe_mem(hipMemcpy(plan[g].lb_boundary_velocity, host_lb_Boundary_velocity, 3*n_lb_boundaries*sizeof(float), hipMemcpyHostToDevice));
    cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(n_lb_boundaries_gpu), &temp, sizeof(int)));

    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

    KERNELCALL(reset_boundaries, dim_grid, threads_per_block, (plan[g].nodes_a, plan[g].nodes_b));

    if (n_lb_boundaries == 0) {
      hipDeviceSynchronize();
      return;
    }
    if(number_of_boundnodes == 0){
      fprintf(stderr, "WARNING: boundary cmd executed but no boundary node found!\n");
    } else{
      int threads_per_block_bound = 64;
      int blocks_per_grid_bound_y = 4;
      int blocks_per_grid_bound_x = (number_of_boundnodes + threads_per_block_bound * blocks_per_grid_bound_y - 1) /(threads_per_block_bound * blocks_per_grid_bound_y);
      dim3 dim_grid_bound = make_uint3(blocks_per_grid_bound_x, blocks_per_grid_bound_y, 1);

      KERNELCALL(init_boundaries, dim_grid_bound, threads_per_block_bound, (boundary_node_list, boundary_index_list, number_of_boundnodes, plan[g].nodes_a, plan[g].nodes_b));
    }

    hipDeviceSynchronize();
  }
}
#endif
/**setup and call extern single node force initialization from the host
 * @param *lbpar_gpu				Pointer to host parameter struct
*/
void lb_reinit_extern_nodeforce_GPU(LB_parameters_gpu *lbpar_gpu){

//FIXME
  LB_TRACE(printf("node %i reinit_extern_nodeforce_GPU gpu %i\n", this_node, lbdevicepar_gpu->gpu_number));
  //begin loop over devices i
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu->gpu_number));
    cuda_check_errors(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
    cuda_check_errors(hipMemcpyToSymbol(HIP_SYMBOL(devpara), lbdevicepar_gpu, sizeof(LB_gpus)));

    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu->number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

    KERNELCALL(reinit_node_force, dim_grid, threads_per_block, (plan[g].node_f));
  }
}
/**setup and call extern single node force initialization from the host
 * @param n_extern_nodeforces			number of nodes on which the external force has to be applied
 * @param *host_extern_nodeforces		Pointer to the host extern node forces
 * @param *lbpar_gpu				Pointer to host parameter struct
*/
void lb_init_extern_nodeforces_GPU(int n_extern_nodeforces, LB_extern_nodeforce_gpu *host_extern_nodeforces, LB_parameters_gpu *lbpar_gpu){
//FIXME
  LB_TRACE(printf("node %i init_extern_nodeforces_GPU gpu %i\n", this_node, lbdevicepar_gpu->gpu_number));

  //begin loop over devices i
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu->gpu_number));
    size_of_extern_nodeforces = n_extern_nodeforces*sizeof(LB_extern_nodeforce_gpu);
    cuda_safe_mem(hipMalloc((void**)&plan[g]extern_nodeforces, size_of_extern_nodeforces));
    hipMemcpy(plan[g].extern_nodeforces, host_extern_nodeforces, size_of_extern_nodeforces, hipMemcpyHostToDevice);

    if(lbpar_gpu->external_force == 0)cuda_safe_mem(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu))); 

    int threads_per_block_exf = 64;
    int blocks_per_grid_exf_y = 4;
    int blocks_per_grid_exf_x = (n_extern_nodeforces + threads_per_block_exf * blocks_per_grid_exf_y - 1) /(threads_per_block_exf * blocks_per_grid_exf_y);
    dim3 dim_grid_exf = make_uint3(blocks_per_grid_exf_x, blocks_per_grid_exf_y, 1);
	
    KERNELCALL(init_extern_nodeforces, dim_grid_exf, threads_per_block_exf, (n_extern_nodeforces, plan[g].extern_nodeforces, plan[g].node_f));
    hipFree(plan[g].extern_nodeforces);
  }
}

/**setup and call particle kernel from the host
*/
void lb_calc_particle_lattice_ia_gpu(){
  if (lbpar_gpu.number_of_particles) {
    //begin loop over devices g
    LB_TRACE(printf("node %i particle_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
    for(int g = 0; g < gpu_n; ++g){
      //set device i
      cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
      /** call of the particle kernel */
      /** values for the particle kernel */
      int threads_per_block_particles = 64;
      int blocks_per_grid_particles_y = 4;
      int blocks_per_grid_particles_x = (lbdevicepar_gpu.number_of_particles + threads_per_block_particles * blocks_per_grid_particles_y - 1)/(threads_per_block_particles * blocks_per_grid_particles_y);
      dim3 dim_grid_particles = make_uint3(blocks_per_grid_particles_x, blocks_per_grid_particles_y, 1);

      KERNELCALL(calc_fluid_particle_ia, dim_grid_particles, threads_per_block_particles, (*plan[g].current_nodes, gpu_get_particle_pointer(), gpu_get_particle_force_pointer(), plan[g].node_f, gpu_get_particle_seed_pointer(), plan[g].device_rho_v));
    }
  }
}

/** setup and call kernel for getting macroscopic fluid values of all nodes
 * @param *host_values struct to save the gpu values
*/
void lb_get_values_GPU(LB_rho_v_pi_gpu *host_values){
//FIXME
  LB_TRACE(printf("node %i get values gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    //printf("current pointer %p\n", plan[g].current_nodes->vd);
    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

    if(lbdevicepar_gpu.number_of_gpus == 1){
      KERNELCALL(get_mesoscopic_values_in_MD_units, dim_grid, threads_per_block, (plan[g].nodes_a, plan[g].print_rho_v_pi, plan[g].device_rho_v ));
      hipMemcpy(host_values, plan[g].print_rho_v_pi, size_of_rho_v_pi, hipMemcpyDeviceToHost);
    }else{
      KERNELCALL(get_mesoscopic_values_in_MD_units_wo_halo, dim_grid, threads_per_block, (plan[g].nodes_a, plan[g].print_rho_v_pi, plan[g].device_rho_v ));
      hipMemcpy(host_values, plan[g].print_rho_v_pi, size_of_rho_v_pi, hipMemcpyDeviceToHost);
    }
  }
}

/** get all the boundary flags for all nodes
 *  @param host_bound_array here go the values of the boundary flag
 */
void lb_get_boundary_flags_GPU(unsigned int* host_bound_array){
  //FIXME 
  LB_TRACE(printf("node %i get_boundary_flags_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    unsigned int* device_bound_array;
    cuda_safe_mem(hipMalloc((void**)&device_bound_array, lbpar_gpu.number_of_nodes*sizeof(unsigned int)));	
    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) / (threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
    if(lbdevicepar_gpu.number_of_gpus ==1){
      cuda_check_errors(hipMalloc((void**)&device_bound_array, lbpar_gpu.number_of_nodes*sizeof(unsigned int)));
      KERNELCALL(lb_get_boundaries, dim_grid, threads_per_block, (*plan[g].current_nodes, device_bound_array));
      hipMemcpy(host_bound_array, device_bound_array, lbpar_gpu.number_of_nodes*sizeof(unsigned int), hipMemcpyDeviceToHost);
    }else{
      cuda_check_errors(hipMalloc((void**)&device_bound_array, lbpar_gpu.number_of_nodes_wo_halo*sizeof(unsigned int)));
      KERNELCALL(lb_get_boundaries_wo_halo, dim_grid, threads_per_block, (*plan[g].current_nodes, device_bound_array));
      hipMemcpy(host_bound_array, device_bound_array, lbpar_gpu.number_of_nodes_wo_halo*sizeof(unsigned int), hipMemcpyDeviceToHost);
    }
    hipFree(device_bound_array);
  }
}

/** setup and call kernel for getting macroscopic fluid values of a single node*/
void lb_print_node_GPU(int single_nodeindex, LB_rho_v_pi_gpu *host_print_values){ 
//FIXME
  LB_TRACE(printf("node %i calc_fluid_mass_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device g
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    LB_rho_v_pi_gpu *device_print_values;
    cuda_safe_mem(hipMalloc((void**)&device_print_values, sizeof(LB_rho_v_pi_gpu)));	
    int threads_per_block_print = 1;
    int blocks_per_grid_print_y = 1;
    int blocks_per_grid_print_x = 1;
    dim3 dim_grid_print = make_uint3(blocks_per_grid_print_x, blocks_per_grid_print_y, 1);

    KERNELCALL(lb_print_node, dim_grid_print, threads_per_block_print, (single_nodeindex, device_print_values, *plan[g].current_nodes, plan[g].device_rho_v));

    hipMemcpy(host_print_values, device_print_values, sizeof(LB_rho_v_pi_gpu), hipMemcpyDeviceToHost);
    hipFree(device_print_values);
  }

}

/** setup and call kernel to calculate the total momentum of the hole fluid
 * @param *mass value of the mass calcutated on the GPU
*/
void lb_calc_fluid_mass_GPU(double* mass){
//FIXME
  LB_TRACE(printf("node %i calc_fluid_mass_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device g
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    float* tot_mass;
    float cpu_mass =  0.f ;
    cuda_safe_mem(hipMalloc((void**)&tot_mass, sizeof(float)));
    hipMemcpy(tot_mass, &cpu_mass, sizeof(float), hipMemcpyHostToDevice);

    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

    KERNELCALL(calc_mass, dim_grid, threads_per_block,(*plan[g].current_nodes, tot_mass));

    hipMemcpy(&cpu_mass, tot_mass, sizeof(float), hipMemcpyDeviceToHost);
  
    hipFree(tot_mass);
    mass[0] = (double)(cpu_mass);
  }
}

/** setup and call kernel to calculate the total momentum of the hole fluid
 *  @param host_mom value of the momentum calcutated on the GPU
 */
void lb_calc_fluid_momentum_GPU(double* host_mom){
//FIXME
  LB_TRACE(printf("node %i calc_fluid_momentum_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device g
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    float* tot_momentum;
    float host_momentum[3] = { 0.f, 0.f, 0.f};
    cuda_safe_mem(hipMalloc((void**)&tot_momentum, 3*sizeof(float)));
    hipMemcpy(tot_momentum, host_momentum, 3*sizeof(float), hipMemcpyHostToDevice);

    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

    KERNELCALL(momentum, dim_grid, threads_per_block,(*plan[g].current_nodes, plan[g].device_rho_v, plan[g].node_f, tot_momentum));
  
    hipMemcpy(host_momentum, tot_momentum, 3*sizeof(float), hipMemcpyDeviceToHost);
  
    hipFree(tot_momentum);
    host_mom[0] = (double)(host_momentum[0]* lbpar_gpu.agrid/lbpar_gpu.tau);
    host_mom[1] = (double)(host_momentum[1]* lbpar_gpu.agrid/lbpar_gpu.tau);
    host_mom[2] = (double)(host_momentum[2]* lbpar_gpu.agrid/lbpar_gpu.tau);
  }
}


/** setup and call kernel to calculate the temperature of the hole fluid
 *  @param host_temp value of the temperatur calcutated on the GPU
*/
void lb_calc_fluid_temperature_GPU(double* host_temp){
//FIXME
  LB_TRACE(printf("node %i calc_fluid_temperature_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device g
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    float host_jsquared = 0.f;
    float* device_jsquared;
    cuda_safe_mem(hipMalloc((void**)&device_jsquared, sizeof(float)));
    hipMemcpy(device_jsquared, &host_jsquared, sizeof(float), hipMemcpyHostToDevice);

    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

    KERNELCALL(temperature, dim_grid, threads_per_block,(*plan[g].current_nodes, device_jsquared));

    hipMemcpy(&host_jsquared, device_jsquared, sizeof(float), hipMemcpyDeviceToHost);
    // TODO: check that temperature calculation is properly implemented for shanchen
    *host_temp=0;
  #pragma unroll
    for(int ii=0;ii<LB_COMPONENTS;++ii) { 
        *host_temp += (double)(host_jsquared*1./(3.f*lbpar_gpu.rho[ii]*lbpar_gpu.dim_x*lbpar_gpu.dim_y*lbpar_gpu.dim_z*lbpar_gpu.tau*lbpar_gpu.tau*lbpar_gpu.agrid));
    }
  }
}


#ifdef SHANCHEN
void lb_calc_shanchen_GPU(){
    //FIXME
  LB_TRACE(printf("node %i calc_shanchen_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device g
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

  KERNELCALL(lb_shanchen_GPU, dim_grid, threads_per_block,(*plan[g].current_nodes, plan[g].node_f));
  }
}

#endif // SHANCHEN






/** setup and call kernel for getting macroscopic fluid values of all nodes
 * @param *host_values struct to save the gpu values
*/
void lb_save_checkpoint_GPU(float *host_checkpoint_vd, unsigned int *host_checkpoint_seed, unsigned int *host_checkpoint_boundary, float *host_checkpoint_force){
//FIXME
  LB_TRACE(printf("node %i save cheackpoint gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    //printf("current pointer %p\n", plan[g].current_nodes->vd);
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    hipMemcpy(host_checkpoint_vd, plan[g].current_nodes->vd, size_of_nodes_gpu, hipMemcpyDeviceToHost);
    hipMemcpy(host_checkpoint_seed, plan[g].current_nodes->seed, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(host_checkpoint_boundary, plan[g].current_nodes->boundary, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyDeviceToHost);
    hipMemcpy(host_checkpoint_force, plan[g].node_f.force, lbpar_gpu.number_of_nodes * 3 * sizeof(float), hipMemcpyDeviceToHost);
  }
}
/** setup and call kernel for setting macroscopic fluid values of all nodes
 * @param *host_values struct to set stored values
*/
void lb_load_checkpoint_GPU(float *host_checkpoint_vd, unsigned int *host_checkpoint_seed, unsigned int *host_checkpoint_boundary, float *host_checkpoint_force){
//FIXME
  LB_TRACE(printf("node %i load cheackpoint gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    hipMemcpy(plan[g].current_nodes->vd, host_checkpoint_vd, size_of_nodes_gpu, hipMemcpyHostToDevice);
      plan[g].intflag = 1;
    hipMemcpy(plan[g].current_nodes->seed, host_checkpoint_seed, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(plan[g].current_nodes->boundary, host_checkpoint_boundary, lbpar_gpu.number_of_nodes * sizeof(unsigned int), hipMemcpyHostToDevice);
    hipMemcpy(plan[g].node_f.force, host_checkpoint_force, lbpar_gpu.number_of_nodes * 3 * sizeof(float), hipMemcpyHostToDevice);
  }
}


/** setup and call kernel to get the boundary flag of a single node
 *  @param single_nodeindex number of the node to get the flag for
 *  @param host_flag her goes the value of the boundary flag
 */
void lb_get_boundary_flag_GPU(int single_nodeindex, unsigned int* host_flag){
  //FIXME
  LB_TRACE(printf("node %i get_bounday_flag_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    unsigned int* device_flag;
    cuda_safe_mem(hipMalloc((void**)&device_flag, sizeof(unsigned int)));	
    int threads_per_block_flag = 1;
    int blocks_per_grid_flag_y = 1;
    int blocks_per_grid_flag_x = 1;
    dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);
//TODO
    KERNELCALL(lb_get_boundary_flag, dim_grid_flag, threads_per_block_flag, (single_nodeindex, device_flag, *plan[g].current_nodes));

    hipMemcpy(host_flag, device_flag, sizeof(unsigned int), hipMemcpyDeviceToHost);

    hipFree(device_flag);
  }
}

/** set the density at a single node
 *  @param single_nodeindex the node to set the velocity for 
 *  @param host_velocity the velocity to set
 */
void lb_set_node_rho_GPU(int single_nodeindex, float* host_rho){
  //FIXME
  LB_TRACE(printf("node %i set_node_rho_GPU gpu %i \n",this_node, lbdevicepar_gpu->gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    float* device_rho;
    cuda_safe_mem(hipMalloc((void**)&device_rho, LB_COMPONENTS*sizeof(float)));	
    hipMemcpy(device_rho, host_rho, LB_COMPONENTS*sizeof(float), hipMemcpyHostToDevice);
    int threads_per_block_flag = 1;
    int blocks_per_grid_flag_y = 1;
    int blocks_per_grid_flag_x = 1;
    dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);
    //TODO
    KERNELCALL(set_rho, dim_grid_flag, threads_per_block_flag, (*plan[g].current_nodes, plan[g].device_rho_v, single_nodeindex, device_rho)); 
    hipFree(device_rho);
  }
}

/** set the net velocity at a single node
 *  @param single_nodeindex the node to set the velocity for 
 *  @param host_velocity the velocity to set
 */
void lb_set_node_velocity_GPU(int single_nodeindex, float* host_velocity){
   
  LB_TRACE(printf("node %i set_node_velocity_GPU gpu %i \n",this_node, lbdevicepar_gpu.gpu_number));
  //begin loop over devices g
  for(int g = 0; g < gpu_n; ++g){
    //set device i
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    float* device_velocity;
    cuda_safe_mem(hipMalloc((void**)&device_velocity, 3*sizeof(float)));	
    hipMemcpy(device_velocity, host_velocity, 3*sizeof(float), hipMemcpyHostToDevice);
    int threads_per_block_flag = 1;
    int blocks_per_grid_flag_y = 1;
    int blocks_per_grid_flag_x = 1;
    dim3 dim_grid_flag = make_uint3(blocks_per_grid_flag_x, blocks_per_grid_flag_y, 1);

    KERNELCALL(set_u_equilibrium, dim_grid_flag, threads_per_block_flag, (*plan[g].current_nodes, single_nodeindex, device_velocity)); 
    hipFree(device_velocity);
  }

}

/** reinit of params 
 * @param *lbpar_gpu struct containing the paramters of the fluid
*/
void reinit_parameters_GPU(LB_parameters_gpu *lbpar_gpu){
  //begin loop over devices g
  LB_TRACE(printf("node %i reinit_parameters_GPU gpu %i \n",this_node, lbdevicepar_gpu->gpu_number));
  //printf("parameter gpu_n %i\n", gpu_n);
  for(int g = 0; g < gpu_n; ++g){
    //set device g
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu->gpu_number));
 
    /**write parameters in const memory*/
    cuda_check_errors(hipMemcpyToSymbol(HIP_SYMBOL(para), lbpar_gpu, sizeof(LB_parameters_gpu)));
    cuda_check_errors(hipMemcpyToSymbol(HIP_SYMBOL(devpara), lbdevicepar_gpu, sizeof(LB_gpus)));
  }
}
#if 0
/**integration kernel for the lb gpu fluid update called from host */
void lb_integrate_GPU(){
  
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);

#ifdef LB_BOUNDARIES_GPU
  if (n_lb_boundaries > 0) 
    cuda_safe_mem(hipMemset	(	LB_boundary_force, 0, 3*n_lb_boundaries*sizeof(float)));
#endif


  /**call of fluid step*/
  /* NOTE: if pi is needed at every integration step, one should call an extended version 
           of the integrate kernel, or pass also device_rho_v_pi and make sure that either 
           it or device_rho_v are NULL depending on extended_values_flag */ 
  if (intflag == 1){
    KERNELCALL(integrate, dim_grid, threads_per_block, (nodes_a, nodes_b, device_rho_v, node_f));
    current_nodes = &nodes_b;
#ifdef LB_BOUNDARIES_GPU		

    if (n_lb_boundaries > 0) {
        KERNELCALL(bb_read, dim_grid, threads_per_block, (nodes_a, nodes_b, LB_boundary_velocity, LB_boundary_force));
      }
#endif
    intflag = 0;
  }
  else{
    KERNELCALL(integrate, dim_grid, threads_per_block, (nodes_b, nodes_a, device_rho_v, node_f));
    current_nodes = &nodes_a;
#ifdef LB_BOUNDARIES_GPU		

    if (n_lb_boundaries > 0) {
      KERNELCALL(bb_read, dim_grid, threads_per_block, (nodes_b, nodes_a, LB_boundary_velocity, LB_boundary_force));
    }
#endif
    intflag = 1;
  }             
}
#endif
void lb_barrier_GPU(){

  LB_TRACE(printf("node %i barrier_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  for(int g = 0; g < gpu_n; ++g){
    //set device g
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    printf("node %i gpu number %i\n", this_node, lbdevicepar_gpu.gpu_number);
    cuda_check_errors(hipDeviceSynchronize());
  }

}

void lb_send_recv_buffer_GPU(){

  LB_TRACE(printf("node %i send_recv_buffer_GPU gpu %i\n", this_node, lbdevicepar_gpu.gpu_number));
  for(int g = 0; g < gpu_n; ++g){
    //set device g
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    lb_send_recv_buffer(plan[g].send_buffer_d, plan[g].recv_buffer_d);
  }

}
//FIXME make consistent naming
void lb_gpu_get_boundary_forces(double* forces) {
#ifdef LB_BOUNDARIES_GPU
  float* temp = (float*) malloc(3*n_lb_boundaries*sizeof(float));
  cuda_safe_mem(hipMemcpy(temp, LB_boundary_force, 3*n_lb_boundaries*sizeof(float), hipMemcpyDeviceToHost));
  for (int i =0; i<3*n_lb_boundaries; i++) {
    forces[i]=(double)temp[i];
  }
  free(temp);
#endif
}
/**integration kernel for the lb gpu fluid update called from host */
void lb_integrate_multigpu_GPU(){
  //begin loop over devices g
  //printf("integrate gpu_n %i\n", gpu_n);
  /** values for the kernel call */
  int threads_per_block = 64;
  int blocks_per_grid_y = 4;
  int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
  dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
  LB_TRACE(printf("node %i integrate_GPU gpu_number %i\n", this_node, lbdevicepar_gpu.gpu_number));
  for(int g = 0; g < gpu_n; ++g){
    //set device g
    cuda_check_errors(hipSetDevice(lbdevicepar_gpu.gpu_number));
    
    /**call of fluid step*/
    if (plan[g].intflag == 1){
      //printf("current pointer %p nodes a %p nodes b %p\n", plan[g].current_nodes, &plan[g].nodes_a, &plan[g].nodes_b);
      KERNELCALL(integrate, dim_grid, threads_per_block, (plan[g].nodes_a, plan[g].nodes_b, plan[g].device_values, plan[g].node_f, plan[g].send_buffer_d, &gpu_n));
      plan[g].current_nodes = &plan[g].nodes_b;
     // printf("current pointer %p nodes b %p\n", plan[g].current_nodes, &plan[g].nodes_b);
      plan[g].intflag = 0;
    }else{
      KERNELCALL(integrate, dim_grid, threads_per_block, (plan[g].nodes_b, plan[g].nodes_a, plan[g].device_values, plan[g].node_f, plan[g].send_buffer_d, &gpu_n));
      plan[g].current_nodes = &plan[g].nodes_a;
      //hipDeviceSynchronize();
      plan[g].intflag = 1;
    }
  }
}
/**apply bounce back boundaries*/
void lb_bb_bounds_GPU(){
    /** values for the kernel call */
    int threads_per_block = 64;
    int blocks_per_grid_y = 4;
    int blocks_per_grid_x = (lbpar_gpu.number_of_nodes + threads_per_block * blocks_per_grid_y - 1) /(threads_per_block * blocks_per_grid_y);
    dim3 dim_grid = make_uint3(blocks_per_grid_x, blocks_per_grid_y, 1);
  for(int g = 0; g < gpu_n; ++g){
    if (plan[g].intflag == 0){
#ifdef LB_BOUNDARIES_GPU		
      if (n_lb_boundaries > 0) {
        KERNELCALL(bb_read, dim_grid, threads_per_block, (plan[g].nodes_a, plan[g].nodes_b, plan[g].lb_boundary_velocity, plan[g].lb_boundary_force));
      }
      KERNELCALL(bb_write_buffer, dim_grid, threads_per_block, (plan[g].nodes_a, plan[g].nodes_b, plan[g].send_buffer_d));
    }else{
      if (n_lb_boundaries > 0) {
        KERNELCALL(bb_read, dim_grid, threads_per_block, (plan[g].nodes_b, plan[g].nodes_a, plan[g].lb_boundary_velocity, plan[g].lb_boundary_force));
      }
      KERNELCALL(bb_write_buffer, dim_grid, threads_per_block, (plan[g].nodes_b, plan[g].nodes_a, plan[g].send_buffer_d));
    }
  }
#endif
}
#endif /* LB_GPU */
